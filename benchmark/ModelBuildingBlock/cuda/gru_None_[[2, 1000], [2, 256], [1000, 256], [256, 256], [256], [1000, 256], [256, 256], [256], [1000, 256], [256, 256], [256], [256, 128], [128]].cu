
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(512) default_function_kernel(float* __restrict__ T_matmul_NN, float* __restrict__ ph, float* __restrict__ ph_1) {
  T_matmul_NN[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k = 0; k < 1000; ++k) {
    T_matmul_NN[((int)threadIdx.x)] = (T_matmul_NN[((int)threadIdx.x)] + (ph[(((((int)threadIdx.x) >> 8) * 1000) + k)] * ph_1[((k * 256) + (((int)threadIdx.x) & 255))]));
  }
}

extern "C" __global__ void __launch_bounds__(256) default_function_kernel_21(float* __restrict__ T_add, float* __restrict__ T_matmul_NN, float* __restrict__ ph) {
  T_add[((int)threadIdx.x)] = (T_matmul_NN[((int)threadIdx.x)] + ph[(((int)threadIdx.x) & 127)]);
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_2(float* __restrict__ T_matmul_NN, float* __restrict__ T_matmul_NN_1) {
  T_matmul_NN[((int)threadIdx.x)] = (T_matmul_NN[((int)threadIdx.x)] + T_matmul_NN_1[((int)threadIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_19(float* __restrict__ T_matmul_NN, float* __restrict__ T_matmul_NN_1) {
  T_matmul_NN[((int)threadIdx.x)] = (T_matmul_NN[((int)threadIdx.x)] + T_matmul_NN_1[((int)threadIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_5(float* __restrict__ T_matmul_NN, float* __restrict__ T_matmul_NN_1, float* __restrict__ ph) {
  T_matmul_NN[((int)threadIdx.x)] = (T_matmul_NN_1[((int)threadIdx.x)] * ph[((int)threadIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_17(float* __restrict__ T_matmul_NN) {
  T_matmul_NN[((int)threadIdx.x)] = tanhf(T_matmul_NN[((int)threadIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_14(float* __restrict__ T_matmul_NN, float* __restrict__ T_matmul_NN_1, float* __restrict__ ph) {
  T_matmul_NN[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k = 0; k < 256; ++k) {
    T_matmul_NN[((int)threadIdx.x)] = (T_matmul_NN[((int)threadIdx.x)] + (T_matmul_NN_1[(((((int)threadIdx.x) >> 8) * 256) + k)] * ph[((k * 256) + (((int)threadIdx.x) & 255))]));
  }
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_16(float* __restrict__ T_matmul_NN, float* __restrict__ ph) {
  T_matmul_NN[((int)threadIdx.x)] = (T_matmul_NN[((int)threadIdx.x)] + ph[(((int)threadIdx.x) & 255)]);
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_4(float* __restrict__ T_matmul_NN) {
  T_matmul_NN[((int)threadIdx.x)] = (1.000000e+00f / (1.000000e+00f + __expf((0.000000e+00f - T_matmul_NN[((int)threadIdx.x)]))));
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_8(float* __restrict__ T_matmul_NN, float* __restrict__ ph, float* __restrict__ ph_1) {
  T_matmul_NN[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k = 0; k < 1000; ++k) {
    T_matmul_NN[((int)threadIdx.x)] = (T_matmul_NN[((int)threadIdx.x)] + (ph[(((((int)threadIdx.x) >> 8) * 1000) + k)] * ph_1[((k * 256) + (((int)threadIdx.x) & 255))]));
  }
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_13(float* __restrict__ T_matmul_NN, float* __restrict__ ph) {
  T_matmul_NN[((int)threadIdx.x)] = (T_matmul_NN[((int)threadIdx.x)] * ph[((int)threadIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_3(float* __restrict__ T_matmul_NN, float* __restrict__ ph) {
  T_matmul_NN[((int)threadIdx.x)] = (T_matmul_NN[((int)threadIdx.x)] + ph[(((int)threadIdx.x) & 255)]);
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_12(float* __restrict__ T_matmul_NN) {
  T_matmul_NN[((int)threadIdx.x)] = (1.000000e+00f / (1.000000e+00f + __expf((0.000000e+00f - T_matmul_NN[((int)threadIdx.x)]))));
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_11(float* __restrict__ T_matmul_NN, float* __restrict__ ph) {
  T_matmul_NN[((int)threadIdx.x)] = (T_matmul_NN[((int)threadIdx.x)] + ph[(((int)threadIdx.x) & 255)]);
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_9(float* __restrict__ T_matmul_NN, float* __restrict__ ph, float* __restrict__ ph_1) {
  T_matmul_NN[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k = 0; k < 256; ++k) {
    T_matmul_NN[((int)threadIdx.x)] = (T_matmul_NN[((int)threadIdx.x)] + (ph[(((((int)threadIdx.x) >> 8) * 256) + k)] * ph_1[((k * 256) + (((int)threadIdx.x) & 255))]));
  }
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_10(float* __restrict__ T_matmul_NN, float* __restrict__ T_matmul_NN_1) {
  T_matmul_NN[((int)threadIdx.x)] = (T_matmul_NN[((int)threadIdx.x)] + T_matmul_NN_1[((int)threadIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_7(float* __restrict__ T_matmul_NN, float* __restrict__ ph, float* __restrict__ ph_1) {
  T_matmul_NN[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k = 0; k < 1000; ++k) {
    T_matmul_NN[((int)threadIdx.x)] = (T_matmul_NN[((int)threadIdx.x)] + (ph[(((((int)threadIdx.x) >> 8) * 1000) + k)] * ph_1[((k * 256) + (((int)threadIdx.x) & 255))]));
  }
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_15(float* __restrict__ T_matmul_NN, float* __restrict__ T_matmul_NN_1) {
  T_matmul_NN[((int)threadIdx.x)] = (T_matmul_NN[((int)threadIdx.x)] + T_matmul_NN_1[((int)threadIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_18(float* __restrict__ T_matmul_NN, float* __restrict__ T_matmul_NN_1) {
  T_matmul_NN[((int)threadIdx.x)] = (T_matmul_NN[((int)threadIdx.x)] * T_matmul_NN_1[((int)threadIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(256) default_function_kernel_20(float* __restrict__ T_matmul_NN, float* __restrict__ T_matmul_NN_1, float* __restrict__ ph) {
  T_matmul_NN[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k = 0; k < 256; ++k) {
    T_matmul_NN[((int)threadIdx.x)] = (T_matmul_NN[((int)threadIdx.x)] + (T_matmul_NN_1[(((((int)threadIdx.x) >> 7) * 256) + k)] * ph[((k * 128) + (((int)threadIdx.x) & 127))]));
  }
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_6(float* __restrict__ T_matmul_NN) {
  T_matmul_NN[((int)threadIdx.x)] = (1.000000e+00f - T_matmul_NN[((int)threadIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_1(float* __restrict__ T_matmul_NN, float* __restrict__ ph, float* __restrict__ ph_1) {
  T_matmul_NN[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k = 0; k < 256; ++k) {
    T_matmul_NN[((int)threadIdx.x)] = (T_matmul_NN[((int)threadIdx.x)] + (ph[(((((int)threadIdx.x) >> 8) * 256) + k)] * ph_1[((k * 256) + (((int)threadIdx.x) & 255))]));
  }
}

