
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_13(float* __restrict__ T_matmul_NN, float* __restrict__ T_transpose) {
  T_matmul_NN[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_matmul_NN[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_37(float* __restrict__ T_strided_slice, float* __restrict__ T_transpose) {
  T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_22(float* __restrict__ T_matmul_NN, float* __restrict__ T_strided_slice, float* __restrict__ T_transpose) {
  T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int h = 0; h < 64; ++h) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (T_matmul_NN[(((((int)blockIdx.x) * 512) + ((((int)threadIdx.x) >> 7) * 64)) + h)] * T_transpose[((((((int)blockIdx.x) >> 4) * 8192) + ((((int)threadIdx.x) & 127) * 64)) + h)]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_39(float* __restrict__ T_multiply, float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 129) {
    T_multiply[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (1.000000e+00f / (1.000000e+00f + __expf((0.000000e+00f - T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]))));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_28(float* __restrict__ T_matmul_NN, float* __restrict__ T_transpose) {
  T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_matmul_NN[(((((((int)blockIdx.x) & 7) * 8192) + ((((int)threadIdx.x) >> 6) * 512)) + ((((int)blockIdx.x) >> 3) * 64)) + (((int)threadIdx.x) & 63))];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_8(float* __restrict__ T_matmul_NN, float* __restrict__ T_transpose, float* __restrict__ ph) {
  T_matmul_NN[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * ph[(((((int)blockIdx.x) & 7) * 1024) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_cast, float* __restrict__ ph) {
  T_cast[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_15(float* __restrict__ T_strided_slice, float* __restrict__ T_transpose) {
  T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_transpose[(((((((int)blockIdx.x) & 7) * 8192) + ((((int)threadIdx.x) >> 6) * 512)) + ((((int)blockIdx.x) >> 3) * 64)) + (((int)threadIdx.x) & 63))];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_40(float* __restrict__ T_multiply, float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 129) {
    T_multiply[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_multiply[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_cast, float* __restrict__ T_multiply) {
  T_multiply[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_cast[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_cast[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_11(float* __restrict__ T_strided_slice, float* __restrict__ T_strided_slice_1, float* __restrict__ T_transpose) {
  T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((32 <= (((int)threadIdx.x) & 63)) ? T_strided_slice[((((((int)blockIdx.x) * 512) + ((((int)threadIdx.x) >> 6) * 32)) + (((int)threadIdx.x) & 63)) - 32)] : T_strided_slice_1[(((((int)blockIdx.x) * 512) + ((((int)threadIdx.x) >> 6) * 32)) + (((int)threadIdx.x) & 63))]);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_19(float* __restrict__ T_strided_slice, float* __restrict__ T_strided_slice_1, float* __restrict__ T_strided_slice_2) {
  T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((32 <= (((int)threadIdx.x) & 63)) ? T_strided_slice_1[((((((int)blockIdx.x) * 512) + ((((int)threadIdx.x) >> 6) * 32)) + (((int)threadIdx.x) & 63)) - 32)] : T_strided_slice_2[(((((int)blockIdx.x) * 512) + ((((int)threadIdx.x) >> 6) * 32)) + (((int)threadIdx.x) & 63))]);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_23(float* __restrict__ T_cast, float* __restrict__ T_strided_slice) {
  T_cast[((int)threadIdx.x)] = -3.402823e+38f;
  for (int k = 0; k < 128; ++k) {
    T_cast[((int)threadIdx.x)] = max(T_cast[((int)threadIdx.x)], T_strided_slice[((((int)threadIdx.x) * 128) + k)]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_9(float* __restrict__ T_strided_slice, float* __restrict__ T_transpose) {
  T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_transpose[((((((int)blockIdx.x) * 2048) + ((((int)threadIdx.x) >> 5) * 64)) + (((int)threadIdx.x) & 31)) + 32)];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_16(float* __restrict__ T_strided_slice, float* __restrict__ T_transpose, float* __restrict__ ph) {
  T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * ph[(((((int)blockIdx.x) & 7) * 1024) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_4(float* __restrict__ T_cast, float* __restrict__ T_cast_1, float* __restrict__ T_multiply_red) {
  T_cast[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((T_cast[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_cast_1[(((int)threadIdx.x) & 511)]) * (1.000000e+00f / sqrtf(((T_multiply_red[((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9))] * 1.953125e-03f) + 1.000000e-05f))));
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_25(float* __restrict__ T_cast, float* __restrict__ T_strided_slice) {
  T_cast[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k = 0; k < 128; ++k) {
    T_cast[((int)threadIdx.x)] = (T_cast[((int)threadIdx.x)] + T_strided_slice[((((int)threadIdx.x) * 128) + k)]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_45(float* __restrict__ T_add, float* __restrict__ T_matmul_NN, float* __restrict__ T_transpose) {
  T_add[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + T_matmul_NN[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_7(float* __restrict__ T_matmul_NN, float* __restrict__ T_transpose) {
  T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_matmul_NN[(((((((int)blockIdx.x) & 7) * 8192) + ((((int)threadIdx.x) >> 6) * 512)) + ((((int)blockIdx.x) >> 3) * 64)) + (((int)threadIdx.x) & 63))];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_6(float* __restrict__ T_matmul_NN, float* __restrict__ T_multiply, float* __restrict__ ph) {
  T_matmul_NN[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k = 0; k < 512; ++k) {
    T_matmul_NN[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_matmul_NN[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (T_multiply[(((((int)blockIdx.x) * 1024) + ((((int)threadIdx.x) >> 9) * 512)) + k)] * ph[((k * 512) + (((int)threadIdx.x) & 511))]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_32(float* __restrict__ T_cast, float* __restrict__ T_transpose) {
  T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + T_cast[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_33(float* __restrict__ T_cast, float* __restrict__ ph) {
  T_cast[((int)threadIdx.x)] = ph[((int)threadIdx.x)];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_20(float* __restrict__ T_strided_slice, float* __restrict__ ph) {
  T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * ph[(((((int)blockIdx.x) & 7) * 1024) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_12(float* __restrict__ T_transpose, float* __restrict__ ph) {
  T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * ph[(((((int)blockIdx.x) & 7) * 1024) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(128) default_function_kernel_3(float* __restrict__ T_multiply, float* __restrict__ T_multiply_red) {
  T_multiply_red[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k2 = 0; k2 < 512; ++k2) {
    T_multiply_red[((int)threadIdx.x)] = (T_multiply_red[((int)threadIdx.x)] + T_multiply[((((int)threadIdx.x) * 512) + k2)]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_10(float* __restrict__ T_strided_slice, float* __restrict__ T_transpose) {
  T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_transpose[(((((int)blockIdx.x) * 2048) + ((((int)threadIdx.x) >> 5) * 64)) + (((int)threadIdx.x) & 31))];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_5(float* __restrict__ T_cast, float* __restrict__ T_multiply) {
  T_multiply[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_cast[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_24(float* __restrict__ T_cast, float* __restrict__ T_strided_slice) {
  T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = __expf((T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] - T_cast[((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 7))]));
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_30(float* __restrict__ T_matmul_NN, float* __restrict__ T_multiply) {
  T_matmul_NN[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_multiply[((((((((int)threadIdx.x) & 511) >> 6) * 8192) + (((int)blockIdx.x) * 128)) + ((((int)threadIdx.x) >> 9) * 64)) + (((int)threadIdx.x) & 63))];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_43(float* __restrict__ T_multiply, float* __restrict__ T_strided_slice, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 129) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k = 0; k < 1376; ++k) {
    if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 129) {
      T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (T_multiply[(((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 5)) / 43) * 1376) + k)] * ph[((k * 1376) + (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) % 1376))]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_17(float* __restrict__ T_strided_slice, float* __restrict__ T_strided_slice_1) {
  T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_strided_slice_1[((((((int)blockIdx.x) * 2048) + ((((int)threadIdx.x) >> 5) * 64)) + (((int)threadIdx.x) & 31)) + 32)];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_29(float* __restrict__ T_multiply, float* __restrict__ T_strided_slice, float* __restrict__ T_transpose) {
  T_multiply[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k = 0; k < 128; ++k) {
    T_multiply[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_multiply[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (T_strided_slice[(((((int)blockIdx.x) * 2048) + ((((int)threadIdx.x) >> 6) * 128)) + k)] * T_transpose[((((((int)blockIdx.x) >> 3) * 8192) + (k * 64)) + (((int)threadIdx.x) & 63))]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_18(float* __restrict__ T_strided_slice, float* __restrict__ T_strided_slice_1) {
  T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_strided_slice_1[(((((int)blockIdx.x) * 2048) + ((((int)threadIdx.x) >> 5) * 64)) + (((int)threadIdx.x) & 31))];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_31(float* __restrict__ T_matmul_NN, float* __restrict__ T_transpose, float* __restrict__ ph) {
  T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k = 0; k < 512; ++k) {
    T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (T_matmul_NN[(((((int)blockIdx.x) * 1024) + ((((int)threadIdx.x) >> 9) * 512)) + k)] * ph[((k * 512) + (((int)threadIdx.x) & 511))]));
  }
}

extern "C" __global__ void __launch_bounds__(512) default_function_kernel_1(float* __restrict__ T_cast, float* __restrict__ ph) {
  T_cast[((int)threadIdx.x)] = ph[((int)threadIdx.x)];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_14(float* __restrict__ T_multiply, float* __restrict__ T_transpose, float* __restrict__ ph) {
  T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k = 0; k < 512; ++k) {
    T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (T_multiply[(((((int)blockIdx.x) * 1024) + ((((int)threadIdx.x) >> 9) * 512)) + k)] * ph[((k * 512) + (((int)threadIdx.x) & 511))]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_21(float* __restrict__ T_strided_slice, float* __restrict__ T_transpose) {
  T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(128) default_function_kernel_35(float* __restrict__ T_multiply, float* __restrict__ T_multiply_red) {
  T_multiply_red[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k2 = 0; k2 < 512; ++k2) {
    T_multiply_red[((int)threadIdx.x)] = (T_multiply_red[((int)threadIdx.x)] + T_multiply[((((int)threadIdx.x) * 512) + k2)]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_27(float* __restrict__ T_matmul_NN, float* __restrict__ T_multiply, float* __restrict__ ph) {
  T_matmul_NN[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int k = 0; k < 512; ++k) {
    T_matmul_NN[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_matmul_NN[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (T_multiply[(((((int)blockIdx.x) * 1024) + ((((int)threadIdx.x) >> 9) * 512)) + k)] * ph[((k * 512) + (((int)threadIdx.x) & 511))]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_34(float* __restrict__ T_multiply, float* __restrict__ T_transpose) {
  T_multiply[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_26(float* __restrict__ T_cast, float* __restrict__ T_strided_slice) {
  T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] / T_cast[((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 7))]);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_44(float* __restrict__ T_matmul_NN, float* __restrict__ T_strided_slice) {
  T_matmul_NN[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_41(float* __restrict__ T_strided_slice, float* __restrict__ T_strided_slice_1, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 129) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k = 0; k < 512; ++k) {
    if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 129) {
      T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (T_strided_slice_1[(((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 5)) / 43) * 512) + k)] * ph[((k * 1376) + (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) % 1376))]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_38(float* __restrict__ T_strided_slice, float* __restrict__ T_strided_slice_1, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 129) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k = 0; k < 512; ++k) {
    if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 129) {
      T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (T_strided_slice_1[(((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 5)) / 43) * 512) + k)] * ph[((k * 1376) + (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) % 1376))]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_42(float* __restrict__ T_multiply, float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 129) {
    T_multiply[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_multiply[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_36(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ T_transpose) {
  T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_cast[(((int)threadIdx.x) & 511)]) * (1.000000e+00f / sqrtf(((T_multiply_red[((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9))] * 1.953125e-03f) + 1.000000e-05f))));
}

