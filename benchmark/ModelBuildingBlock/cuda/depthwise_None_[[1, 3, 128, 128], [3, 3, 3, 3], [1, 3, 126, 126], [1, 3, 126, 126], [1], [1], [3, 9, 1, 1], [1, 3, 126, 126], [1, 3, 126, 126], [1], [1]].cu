
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(3) default_function_kernel_4(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = ph[((int)threadIdx.x)];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_15(float* __restrict__ T_reshape, float* __restrict__ group_conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 11907) {
    group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] - T_reshape[(((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3969)]);
  }
}

extern "C" __global__ void __launch_bounds__(3) default_function_kernel_18(float* __restrict__ T_reshape) {
  T_reshape[((int)threadIdx.x)] = sqrtf(T_reshape[((int)threadIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_23(float* __restrict__ T_reshape, float* __restrict__ group_conv2d_nchw) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 11907) {
    group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + T_reshape[(((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3969)]);
  }
}

extern "C" __global__ void __launch_bounds__(3) default_function_kernel_20(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = ph[0];
}

extern "C" __global__ void __launch_bounds__(3) default_function_kernel_17(float* __restrict__ T_reshape) {
  T_reshape[((int)threadIdx.x)] = (T_reshape[((int)threadIdx.x)] + 1.000000e-05f);
}

extern "C" __global__ void __launch_bounds__(3) default_function_kernel_6(float* __restrict__ T_reshape) {
  T_reshape[((int)threadIdx.x)] = sqrtf(T_reshape[((int)threadIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_21(float* __restrict__ T_reshape, float* __restrict__ group_conv2d_nchw) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 11907) {
    group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_reshape[(((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3969)]);
  }
}

extern "C" __global__ void __launch_bounds__(3) default_function_kernel_22(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = ph[0];
}

extern "C" __global__ void __launch_bounds__(3) default_function_kernel_2(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = ph[((int)threadIdx.x)];
}

extern "C" __global__ void __launch_bounds__(3) default_function_kernel_8(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = ph[0];
}

extern "C" __global__ void __launch_bounds__(3) default_function_kernel_5(float* __restrict__ T_reshape) {
  T_reshape[((int)threadIdx.x)] = (T_reshape[((int)threadIdx.x)] + 1.000000e-05f);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_19(float* __restrict__ T_reshape, float* __restrict__ group_conv2d_nchw) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 11907) {
    group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] / T_reshape[(((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3969)]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_13(float* __restrict__ group_conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 11907) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 3; ++rc) {
    if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 11907) {
      pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (group_conv2d_nchw[((rc * 15876) + (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) % 15876))] * ph[(((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3969) * 9) + rc)]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_9(float* __restrict__ T_reshape, float* __restrict__ group_conv2d_nchw) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 11907) {
    group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_reshape[(((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3969)]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_7(float* __restrict__ T_reshape, float* __restrict__ group_conv2d_nchw) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 11907) {
    group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] / T_reshape[(((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3969)]);
  }
}

extern "C" __global__ void __launch_bounds__(3) default_function_kernel_10(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = ph[0];
}

extern "C" __global__ void __launch_bounds__(3) default_function_kernel_14(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = ph[((int)threadIdx.x)];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_12(float* __restrict__ group_conv2d_nchw) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 11907) {
    group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_11(float* __restrict__ T_reshape, float* __restrict__ group_conv2d_nchw) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 11907) {
    group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + T_reshape[(((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3969)]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ pad_temp, float* __restrict__ ph) {
  pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
}

extern "C" __global__ void __launch_bounds__(3) default_function_kernel_16(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = ph[((int)threadIdx.x)];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_24(float* __restrict__ compute, float* __restrict__ group_conv2d_nchw) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 11907) {
    compute[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ group_conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 11907) {
    group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int ry = 0; ry < 3; ++ry) {
    for (int rx = 0; rx < 3; ++rx) {
      if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 11907) {
        group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[((((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3969) * 16384) + (((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 7938) / 63) * 128)) + (ry * 128)) + rx) + (((((int)blockIdx.x) * 16) + ((int)threadIdx.x)) % 126))] * ph[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3969) * 27) + (ry * 3)) + rx)]));
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ T_reshape, float* __restrict__ group_conv2d_nchw) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 11907) {
    group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (group_conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] - T_reshape[(((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3969)]);
  }
}

