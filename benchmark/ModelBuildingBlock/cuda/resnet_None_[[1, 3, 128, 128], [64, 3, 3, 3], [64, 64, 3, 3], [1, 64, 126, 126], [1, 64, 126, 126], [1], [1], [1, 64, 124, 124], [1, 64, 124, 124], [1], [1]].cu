
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ T_reshape, float* __restrict__ conv2d_nchw) {
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 1024) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) >> 8)) < 3969) {
      conv2d_nchw[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (conv2d_nchw[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] - T_reshape[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) >> 2)) / 3969)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_12(float* __restrict__ conv2d_nchw) {
  for (int i0_i1_fused_i2_fused_i3_fused_outer = 0; i0_i1_fused_i2_fused_i3_fused_outer < 4; ++i0_i1_fused_i2_fused_i3_fused_outer) {
    if ((((i0_i1_fused_i2_fused_i3_fused_outer * 1024) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) >> 8)) < 3969) {
      conv2d_nchw[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = max(conv2d_nchw[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))], 0.000000e+00f);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_19(float* __restrict__ T_reshape, float* __restrict__ conv2d_nchw) {
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 256) + ((int)blockIdx.x)) < 961) {
      conv2d_nchw[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (conv2d_nchw[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] / T_reshape[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 16384) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) >> 4)) / 961)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_13(float* __restrict__ conv2d_nchw, float* __restrict__ conv2d_nchw_1, float* __restrict__ ph) {
  for (int nn_ff_fused_yy_fused_xx_fused_outer = 0; nn_ff_fused_yy_fused_xx_fused_outer < 4; ++nn_ff_fused_yy_fused_xx_fused_outer) {
    if (((nn_ff_fused_yy_fused_xx_fused_outer * 256) + ((int)blockIdx.x)) < 961) {
      conv2d_nchw[(((nn_ff_fused_yy_fused_xx_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = 0.000000e+00f;
    }
    for (int rc = 0; rc < 64; ++rc) {
      for (int ry = 0; ry < 3; ++ry) {
        for (int rx = 0; rx < 3; ++rx) {
          if (((nn_ff_fused_yy_fused_xx_fused_outer * 256) + ((int)blockIdx.x)) < 961) {
            conv2d_nchw[(((nn_ff_fused_yy_fused_xx_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (conv2d_nchw[(((nn_ff_fused_yy_fused_xx_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] + (conv2d_nchw_1[(((((rc * 15876) + ((((((nn_ff_fused_yy_fused_xx_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) >> 2)) % 3844) / 31) * 126)) + (ry * 126)) + rx) + ((((nn_ff_fused_yy_fused_xx_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x)) % 124))] * ph[(((((((((nn_ff_fused_yy_fused_xx_fused_outer * 16384) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) >> 4)) % 61504) / 961) * 576) + (rc * 9)) + (ry * 3)) + rx)]));
          }
        }
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_10(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = ph[0];
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_18(float* __restrict__ T_reshape) {
  T_reshape[((int)threadIdx.x)] = sqrtf(T_reshape[((int)threadIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_6(float* __restrict__ T_reshape) {
  T_reshape[((int)threadIdx.x)] = sqrtf(T_reshape[((int)threadIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_24(float* __restrict__ compute, float* __restrict__ conv2d_nchw) {
  for (int i0_i1_fused_i2_fused_i3_fused_outer = 0; i0_i1_fused_i2_fused_i3_fused_outer < 4; ++i0_i1_fused_i2_fused_i3_fused_outer) {
    if (((i0_i1_fused_i2_fused_i3_fused_outer * 256) + ((int)blockIdx.x)) < 961) {
      compute[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = max(conv2d_nchw[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))], 0.000000e+00f);
    }
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_17(float* __restrict__ T_reshape) {
  T_reshape[((int)threadIdx.x)] = (T_reshape[((int)threadIdx.x)] + 1.000000e-05f);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_9(float* __restrict__ T_reshape, float* __restrict__ conv2d_nchw) {
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 1024) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) >> 8)) < 3969) {
      conv2d_nchw[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (conv2d_nchw[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] * T_reshape[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) >> 2)) / 3969)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_20(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = ph[0];
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_8(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = ph[0];
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_4(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = ph[((int)threadIdx.x)];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_23(float* __restrict__ T_reshape, float* __restrict__ conv2d_nchw) {
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 256) + ((int)blockIdx.x)) < 961) {
      conv2d_nchw[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (conv2d_nchw[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] + T_reshape[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 16384) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) >> 4)) / 961)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_15(float* __restrict__ T_reshape, float* __restrict__ conv2d_nchw) {
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 256) + ((int)blockIdx.x)) < 961) {
      conv2d_nchw[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (conv2d_nchw[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] - T_reshape[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 16384) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) >> 4)) / 961)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = ph[((int)threadIdx.x)];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_11(float* __restrict__ T_reshape, float* __restrict__ conv2d_nchw) {
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 1024) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) >> 8)) < 3969) {
      conv2d_nchw[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (conv2d_nchw[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] + T_reshape[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) >> 2)) / 3969)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_14(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = ph[((int)threadIdx.x)];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  for (int nn_ff_fused_yy_fused_xx_fused_outer = 0; nn_ff_fused_yy_fused_xx_fused_outer < 4; ++nn_ff_fused_yy_fused_xx_fused_outer) {
    if ((((nn_ff_fused_yy_fused_xx_fused_outer * 1024) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) >> 8)) < 3969) {
      conv2d_nchw[(((nn_ff_fused_yy_fused_xx_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = 0.000000e+00f;
    }
    for (int rc = 0; rc < 3; ++rc) {
      for (int ry = 0; ry < 3; ++ry) {
        for (int rx = 0; rx < 3; ++rx) {
          if ((((nn_ff_fused_yy_fused_xx_fused_outer * 1024) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) >> 8)) < 3969) {
            conv2d_nchw[(((nn_ff_fused_yy_fused_xx_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (conv2d_nchw[(((nn_ff_fused_yy_fused_xx_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] + (pad_temp[(((((rc * 16384) + ((((((nn_ff_fused_yy_fused_xx_fused_outer * 131072) + (((int)blockIdx.x) * 512)) + (((int)threadIdx.x) >> 1)) % 7938) / 63) * 128)) + (ry * 128)) + rx) + ((((nn_ff_fused_yy_fused_xx_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x)) % 126))] * ph[((((((((nn_ff_fused_yy_fused_xx_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) >> 2)) / 3969) * 27) + (rc * 9)) + (ry * 3)) + rx)]));
          }
        }
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_5(float* __restrict__ T_reshape) {
  T_reshape[((int)threadIdx.x)] = (T_reshape[((int)threadIdx.x)] + 1.000000e-05f);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_7(float* __restrict__ T_reshape, float* __restrict__ conv2d_nchw) {
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 1024) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) >> 8)) < 3969) {
      conv2d_nchw[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (conv2d_nchw[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] / T_reshape[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) >> 2)) / 3969)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_21(float* __restrict__ T_reshape, float* __restrict__ conv2d_nchw) {
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 256) + ((int)blockIdx.x)) < 961) {
      conv2d_nchw[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (conv2d_nchw[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] * T_reshape[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 16384) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) >> 4)) / 961)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ pad_temp, float* __restrict__ ph) {
  pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_22(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = ph[0];
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_16(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = ph[((int)threadIdx.x)];
}

