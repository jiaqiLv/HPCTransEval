
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ pad_temp, float* __restrict__ pool_max) {
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 12; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    pool_max[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = -3.402823e+38f;
    for (int rv0 = 0; rv0 < 3; ++rv0) {
      for (int rv1 = 0; rv1 < 3; ++rv1) {
        pool_max[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = max(pool_max[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))], pad_temp[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 32) + (((int)blockIdx.x) >> 3)) / 3) * 24960) + (rv0 * 24960)) + (rv1 * 192)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x)) % 24576))]);
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_12(float* __restrict__ conv2d_nhwc) {
  for (int i0_i1_fused_i2_fused_i3_fused_outer = 0; i0_i1_fused_i2_fused_i3_fused_outer < 6; ++i0_i1_fused_i2_fused_i3_fused_outer) {
    conv2d_nhwc[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = max(conv2d_nhwc[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_13(float* __restrict__ conv2d_nhwc, float* __restrict__ pad_temp) {
  for (int i0_i1_fused_i2_fused_i3_fused_outer = 0; i0_i1_fused_i2_fused_i3_fused_outer < 7; ++i0_i1_fused_i2_fused_i3_fused_outer) {
    if ((((i0_i1_fused_i2_fused_i3_fused_outer * 2048) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) >> 7)) < 12675) {
      pad_temp[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (((((195 <= (((i0_i1_fused_i2_fused_i3_fused_outer * 4096) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) >> 6))) && ((((i0_i1_fused_i2_fused_i3_fused_outer * 4096) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) >> 6)) < 25155)) && (3 <= ((((i0_i1_fused_i2_fused_i3_fused_outer * 8192) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) >> 5)) % 390))) && (((((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x)) % 12480) < 12384)) ? conv2d_nhwc[(((((((i0_i1_fused_i2_fused_i3_fused_outer * 4096) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) >> 6)) / 195) * 12288) + ((((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x)) % 12480)) - 12384)] : 0.000000e+00f);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_4(float* __restrict__ pad_temp, float* __restrict__ ph) {
  for (int i0_i1_fused_i2_fused_i3_fused_outer = 0; i0_i1_fused_i2_fused_i3_fused_outer < 12; ++i0_i1_fused_i2_fused_i3_fused_outer) {
    pad_temp[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = ph[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ pad_temp, float* __restrict__ pool_max) {
  for (int i0_i1_fused_i2_fused_i3_fused_outer = 0; i0_i1_fused_i2_fused_i3_fused_outer < 2; ++i0_i1_fused_i2_fused_i3_fused_outer) {
    pool_max[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = max(pad_temp[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_10(float* __restrict__ pad_temp, float* __restrict__ ph) {
  for (int i0_i1_fused_i2_fused_i3_fused_outer = 0; i0_i1_fused_i2_fused_i3_fused_outer < 12; ++i0_i1_fused_i2_fused_i3_fused_outer) {
    pad_temp[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = ph[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_7(float* __restrict__ conv2d_nhwc, float* __restrict__ pad_temp) {
  for (int i0_i1_fused_i2_fused_i3_fused_outer = 0; i0_i1_fused_i2_fused_i3_fused_outer < 2; ++i0_i1_fused_i2_fused_i3_fused_outer) {
    if ((((i0_i1_fused_i2_fused_i3_fused_outer * 1024) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) >> 8)) < 1089) {
      pad_temp[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (((((33 <= (((i0_i1_fused_i2_fused_i3_fused_outer * 2048) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) >> 7))) && ((((i0_i1_fused_i2_fused_i3_fused_outer * 2048) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) >> 7)) < 2145)) && (1 <= ((((i0_i1_fused_i2_fused_i3_fused_outer * 8192) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) >> 5)) % 66))) && (((((i0_i1_fused_i2_fused_i3_fused_outer * 16384) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) >> 4)) % 132) < 130)) ? conv2d_nhwc[((((((((i0_i1_fused_i2_fused_i3_fused_outer * 4096) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) >> 6)) / 33) * 2048) + (((((i0_i1_fused_i2_fused_i3_fused_outer * 16384) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) >> 4)) % 132) * 16)) + (((int)threadIdx.x) & 15)) - 4128)] : 0.000000e+00f);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_14(float* __restrict__ conv2d_nhwc, float* __restrict__ pad_temp, float* __restrict__ ph) {
  for (int nn_yy_fused_xx_fused_ff_fused_outer = 0; nn_yy_fused_xx_fused_ff_fused_outer < 8; ++nn_yy_fused_xx_fused_ff_fused_outer) {
    conv2d_nhwc[(((nn_yy_fused_xx_fused_ff_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = 0.000000e+00f;
    for (int ry = 0; ry < 3; ++ry) {
      for (int rx = 0; rx < 3; ++rx) {
        for (int rc = 0; rc < 96; ++rc) {
          conv2d_nhwc[(((nn_yy_fused_xx_fused_ff_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (conv2d_nhwc[(((nn_yy_fused_xx_fused_ff_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] + (pad_temp[(((((((nn_yy_fused_xx_fused_ff_fused_outer * 199680) + ((((int)blockIdx.x) >> 4) * 12480)) + (ry * 12480)) + ((((int)blockIdx.x) & 15) * 768)) + ((((int)threadIdx.x) >> 7) * 96)) + (rx * 96)) + rc)] * ph[((((ry * 36864) + (rx * 12288)) + (rc * 128)) + (((int)threadIdx.x) & 127))]));
        }
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_19(float* __restrict__ T_concat, float* __restrict__ conv2d_nhwc, float* __restrict__ conv2d_nhwc_1, float* __restrict__ conv2d_nhwc_2, float* __restrict__ pool_max) {
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 16; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    T_concat[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = ((224 <= (((int)threadIdx.x) & 255)) ? pool_max[(((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 32768) + (((int)blockIdx.x) * 128)) + ((((int)threadIdx.x) >> 8) * 32)) + (((int)threadIdx.x) & 255)) - 224)] : ((192 <= (((int)threadIdx.x) & 255)) ? conv2d_nhwc[(((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 32768) + (((int)blockIdx.x) * 128)) + ((((int)threadIdx.x) >> 8) * 32)) + (((int)threadIdx.x) & 255)) - 192)] : ((64 <= (((int)threadIdx.x) & 255)) ? conv2d_nhwc_1[(((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 131072) + (((int)blockIdx.x) * 512)) + ((((int)threadIdx.x) >> 8) * 128)) + (((int)threadIdx.x) & 255)) - 64)] : conv2d_nhwc_2[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((((int)threadIdx.x) >> 8) * 64)) + (((int)threadIdx.x) & 255))])));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_9(float* __restrict__ conv2d_nhwc) {
  for (int i0_i1_fused_i2_fused_i3_fused_outer = 0; i0_i1_fused_i2_fused_i3_fused_outer < 2; ++i0_i1_fused_i2_fused_i3_fused_outer) {
    conv2d_nhwc[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = max(conv2d_nhwc[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_5(float* __restrict__ conv2d_nhwc, float* __restrict__ pad_temp, float* __restrict__ ph) {
  conv2d_nhwc[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int rc = 0; rc < 192; ++rc) {
    conv2d_nhwc[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv2d_nhwc[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[(((((int)blockIdx.x) * 12288) + ((((int)threadIdx.x) >> 4) * 192)) + rc)] * ph[((rc * 16) + (((int)threadIdx.x) & 15))]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_6(float* __restrict__ conv2d_nhwc) {
  conv2d_nhwc[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nhwc[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_8(float* __restrict__ conv2d_nhwc, float* __restrict__ pad_temp, float* __restrict__ ph) {
  for (int nn_yy_fused_xx_fused_ff_fused_outer = 0; nn_yy_fused_xx_fused_ff_fused_outer < 2; ++nn_yy_fused_xx_fused_ff_fused_outer) {
    conv2d_nhwc[(((nn_yy_fused_xx_fused_ff_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = 0.000000e+00f;
    for (int ry = 0; ry < 5; ++ry) {
      for (int rx = 0; rx < 5; ++rx) {
        for (int rc = 0; rc < 16; ++rc) {
          conv2d_nhwc[(((nn_yy_fused_xx_fused_ff_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (conv2d_nhwc[(((nn_yy_fused_xx_fused_ff_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] + (pad_temp[(((((((nn_yy_fused_xx_fused_ff_fused_outer * 135168) + ((((int)blockIdx.x) >> 2) * 2112)) + (ry * 2112)) + ((((int)blockIdx.x) & 3) * 512)) + ((((int)threadIdx.x) >> 5) * 16)) + (rx * 16)) + rc)] * ph[((((ry * 2560) + (rx * 512)) + (rc * 32)) + (((int)threadIdx.x) & 31))]));
        }
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_17(float* __restrict__ conv2d_nhwc, float* __restrict__ pad_temp, float* __restrict__ ph) {
  for (int nn_yy_fused_xx_fused_ff_fused_outer = 0; nn_yy_fused_xx_fused_ff_fused_outer < 4; ++nn_yy_fused_xx_fused_ff_fused_outer) {
    conv2d_nhwc[(((nn_yy_fused_xx_fused_ff_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = 0.000000e+00f;
    for (int rc = 0; rc < 192; ++rc) {
      conv2d_nhwc[(((nn_yy_fused_xx_fused_ff_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (conv2d_nhwc[(((nn_yy_fused_xx_fused_ff_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] + (pad_temp[((((nn_yy_fused_xx_fused_ff_fused_outer * 786432) + (((int)blockIdx.x) * 3072)) + ((((int)threadIdx.x) >> 6) * 192)) + rc)] * ph[((rc * 64) + (((int)threadIdx.x) & 63))]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_16(float* __restrict__ pad_temp, float* __restrict__ ph) {
  for (int i0_i1_fused_i2_fused_i3_fused_outer = 0; i0_i1_fused_i2_fused_i3_fused_outer < 12; ++i0_i1_fused_i2_fused_i3_fused_outer) {
    pad_temp[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = ph[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_18(float* __restrict__ conv2d_nhwc) {
  for (int i0_i1_fused_i2_fused_i3_fused_outer = 0; i0_i1_fused_i2_fused_i3_fused_outer < 4; ++i0_i1_fused_i2_fused_i3_fused_outer) {
    conv2d_nhwc[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = max(conv2d_nhwc[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_11(float* __restrict__ conv2d_nhwc, float* __restrict__ pad_temp, float* __restrict__ ph) {
  for (int nn_yy_fused_xx_fused_ff_fused_outer = 0; nn_yy_fused_xx_fused_ff_fused_outer < 6; ++nn_yy_fused_xx_fused_ff_fused_outer) {
    conv2d_nhwc[(((nn_yy_fused_xx_fused_ff_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = 0.000000e+00f;
    for (int rc = 0; rc < 192; ++rc) {
      conv2d_nhwc[(((nn_yy_fused_xx_fused_ff_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (conv2d_nhwc[(((nn_yy_fused_xx_fused_ff_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] + (pad_temp[((((((nn_yy_fused_xx_fused_ff_fused_outer * 8192) + (((int)blockIdx.x) * 32)) + (((int)threadIdx.x) >> 5)) / 3) * 192) + rc)] * ph[((rc * 96) + ((((nn_yy_fused_xx_fused_ff_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x)) % 96))]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_15(float* __restrict__ conv2d_nhwc) {
  for (int i0_i1_fused_i2_fused_i3_fused_outer = 0; i0_i1_fused_i2_fused_i3_fused_outer < 8; ++i0_i1_fused_i2_fused_i3_fused_outer) {
    conv2d_nhwc[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = max(conv2d_nhwc[(((i0_i1_fused_i2_fused_i3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ pad_temp, float* __restrict__ ph) {
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 13; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 1024) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.x) >> 8)) < 12675) {
      pad_temp[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (((((195 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 2048) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) >> 7))) && ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 2048) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) >> 7)) < 25155)) && (3 <= ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 4096) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) >> 6)) % 390))) && (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x)) % 24960) < 24768)) ? ph[(((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 2048) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) >> 7)) / 195) * 24576) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x)) % 24960)) - 24768)] : -3.402823e+38f);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ pad_temp, float* __restrict__ ph, float* __restrict__ pool_max) {
  for (int nn_yy_fused_xx_fused_ff_fused_outer = 0; nn_yy_fused_xx_fused_ff_fused_outer < 2; ++nn_yy_fused_xx_fused_ff_fused_outer) {
    pad_temp[(((nn_yy_fused_xx_fused_ff_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = 0.000000e+00f;
    for (int rc = 0; rc < 192; ++rc) {
      pad_temp[(((nn_yy_fused_xx_fused_ff_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (pad_temp[(((nn_yy_fused_xx_fused_ff_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] + (pool_max[((((nn_yy_fused_xx_fused_ff_fused_outer * 1572864) + (((int)blockIdx.x) * 6144)) + ((((int)threadIdx.x) >> 5) * 192)) + rc)] * ph[((rc * 32) + (((int)threadIdx.x) & 31))]));
    }
  }
}

