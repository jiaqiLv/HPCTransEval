
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_18(float* __restrict__ conv1d_ncw) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (1.000000e+00f / (1.000000e+00f + __expf((0.000000e+00f - conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]))));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_42(float* __restrict__ conv1d_ncw, float* __restrict__ conv1d_ncw_1) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = conv1d_ncw_1[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_39(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 128; ++rc) {
    for (int ry = 0; ry < 3; ++ry) {
      if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
        conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[(((rc * 116) + (ry * 8)) + (((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 100))] * ph[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 384) + (rc * 3)) + ry)]));
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 128; ++rc) {
    for (int ry = 0; ry < 3; ++ry) {
      if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
        conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[(((rc * 102) + ry) + (((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 100))] * ph[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 384) + (rc * 3)) + ry)]));
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_5(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 8)) < 51) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((1 <= (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 1)) % 51)) ? conv1d_ncw[((((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) / 51) * 100) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 102)) - 2)] : 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_41(float* __restrict__ conv1d_ncw, float* __restrict__ conv1d_ncw_1) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + conv1d_ncw_1[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_25(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 128; ++rc) {
    for (int ry = 0; ry < 3; ++ry) {
      if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
        conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[(((rc * 108) + (ry * 4)) + (((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 100))] * ph[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 384) + (rc * 3)) + ry)]));
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_27(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 27) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((8 <= (((((int)blockIdx.x) * 52) + ((int)threadIdx.x)) % 108)) ? conv1d_ncw[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 27) * 100) + (((((int)blockIdx.x) * 52) + ((int)threadIdx.x)) % 108)) - 8)] : 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_30(float* __restrict__ conv1d_ncw, float* __restrict__ conv1d_ncw_1) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + conv1d_ncw_1[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_35(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 29) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((16 <= (((((int)blockIdx.x) * 96) + ((int)threadIdx.x)) % 116)) ? conv1d_ncw[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 29) * 100) + (((((int)blockIdx.x) * 96) + ((int)threadIdx.x)) % 116)) - 16)] : 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_17(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 128; ++rc) {
    for (int ry = 0; ry < 3; ++ry) {
      if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
        conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[(((rc * 104) + (ry * 2)) + (((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 100))] * ph[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 384) + (rc * 3)) + ry)]));
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_19(float* __restrict__ conv1d_ncw, float* __restrict__ conv1d_ncw_1) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + conv1d_ncw_1[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_15(float* __restrict__ conv1d_ncw) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = tanhf(conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_24(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 27) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((8 <= (((((int)blockIdx.x) * 52) + ((int)threadIdx.x)) % 108)) ? conv1d_ncw[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 27) * 100) + (((((int)blockIdx.x) * 52) + ((int)threadIdx.x)) % 108)) - 8)] : 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_28(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 128; ++rc) {
    for (int ry = 0; ry < 3; ++ry) {
      if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
        conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[(((rc * 108) + (ry * 4)) + (((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 100))] * ph[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 384) + (rc * 3)) + ry)]));
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_6(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 128; ++rc) {
    for (int ry = 0; ry < 3; ++ry) {
      if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
        conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[(((rc * 102) + ry) + (((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 100))] * ph[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 384) + (rc * 3)) + ry)]));
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_12(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_23(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_40(float* __restrict__ conv1d_ncw) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (1.000000e+00f / (1.000000e+00f + __expf((0.000000e+00f - conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]))));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 8)) < 51) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((1 <= (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 1)) % 51)) ? conv1d_ncw[((((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) / 51) * 100) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 102)) - 2)] : 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_26(float* __restrict__ conv1d_ncw) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = tanhf(conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_11(float* __restrict__ conv1d_ncw, float* __restrict__ conv1d_ncw_1, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 128; ++rc) {
    if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
      conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv1d_ncw_1[((rc * 100) + (((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 100))] * ph[(((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 64) + rc)]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_7(float* __restrict__ conv1d_ncw) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (1.000000e+00f / (1.000000e+00f + __expf((0.000000e+00f - conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]))));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_21(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 128; ++rc) {
    if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
      pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv1d_ncw[((rc * 100) + (((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 100))] * ph[(((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 128) + rc)]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_34(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_46(float* __restrict__ T_concat, float* __restrict__ conv1d_ncw, float* __restrict__ conv1d_ncw_1, float* __restrict__ conv1d_ncw_2, float* __restrict__ conv1d_ncw_3) {
  T_concat[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((75 <= ((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9))) ? conv1d_ncw[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) - 38400)] : ((25 <= ((int)blockIdx.x)) ? conv1d_ncw_1[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) - 25600)] : ((25 <= ((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9))) ? conv1d_ncw_2[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) - 12800)] : conv1d_ncw_3[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))])));
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_4(float* __restrict__ conv1d_ncw) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = tanhf(conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_33(float* __restrict__ conv1d_ncw, float* __restrict__ conv1d_ncw_1, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 128; ++rc) {
    if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
      conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv1d_ncw_1[((rc * 100) + (((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 100))] * ph[(((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 64) + rc)]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_14(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 128; ++rc) {
    for (int ry = 0; ry < 3; ++ry) {
      if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
        conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[(((rc * 104) + (ry * 2)) + (((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 100))] * ph[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 384) + (rc * 3)) + ry)]));
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_45(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_16(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp) {
  pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((1 <= (((((int)blockIdx.x) * 22) + (((int)threadIdx.x) >> 2)) % 26)) ? conv1d_ncw[((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 13) * 100) + (((((int)blockIdx.x) * 88) + ((int)threadIdx.x)) % 104)) - 4)] : 0.000000e+00f);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_8(float* __restrict__ conv1d_ncw, float* __restrict__ conv1d_ncw_1) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + conv1d_ncw_1[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_32(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 128; ++rc) {
    if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
      pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv1d_ncw[((rc * 100) + (((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 100))] * ph[(((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 128) + rc)]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_43(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 128; ++rc) {
    if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
      pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv1d_ncw[((rc * 100) + (((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 100))] * ph[(((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 128) + rc)]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_20(float* __restrict__ conv1d_ncw, float* __restrict__ conv1d_ncw_1) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = conv1d_ncw_1[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 7)) < 51) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((1 <= (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 1)) % 51)) ? ph[((((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) / 51) * 100) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 102)) - 2)] : 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_38(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 29) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((16 <= (((((int)blockIdx.x) * 96) + ((int)threadIdx.x)) % 116)) ? conv1d_ncw[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 29) * 100) + (((((int)blockIdx.x) * 96) + ((int)threadIdx.x)) % 116)) - 16)] : 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_44(float* __restrict__ conv1d_ncw, float* __restrict__ conv1d_ncw_1, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 128; ++rc) {
    if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
      conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv1d_ncw_1[((rc * 100) + (((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 100))] * ph[(((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 64) + rc)]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_29(float* __restrict__ conv1d_ncw) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (1.000000e+00f / (1.000000e+00f + __expf((0.000000e+00f - conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]))));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_31(float* __restrict__ conv1d_ncw, float* __restrict__ conv1d_ncw_1) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = conv1d_ncw_1[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_37(float* __restrict__ conv1d_ncw) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = tanhf(conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_36(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 128; ++rc) {
    for (int ry = 0; ry < 3; ++ry) {
      if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
        conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[(((rc * 116) + (ry * 8)) + (((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 100))] * ph[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 384) + (rc * 3)) + ry)]));
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_9(float* __restrict__ conv1d_ncw, float* __restrict__ conv1d_ncw_1) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = conv1d_ncw_1[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_10(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 128; ++rc) {
    if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
      pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv1d_ncw[((rc * 100) + (((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 100))] * ph[(((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 128) + rc)]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 64; ++rc) {
    for (int ry = 0; ry < 3; ++ry) {
      if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
        conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[(((rc * 102) + ry) + (((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 100))] * ph[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 192) + (rc * 3)) + ry)]));
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_13(float* __restrict__ conv1d_ncw, float* __restrict__ pad_temp) {
  pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((1 <= (((((int)blockIdx.x) * 22) + (((int)threadIdx.x) >> 2)) % 26)) ? conv1d_ncw[((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 13) * 100) + (((((int)blockIdx.x) * 88) + ((int)threadIdx.x)) % 104)) - 4)] : 0.000000e+00f);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_22(float* __restrict__ conv1d_ncw, float* __restrict__ conv1d_ncw_1, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
    conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 128; ++rc) {
    if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 9)) < 25) {
      conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv1d_ncw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv1d_ncw_1[((rc * 100) + (((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 100))] * ph[(((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 64) + rc)]));
    }
  }
}

