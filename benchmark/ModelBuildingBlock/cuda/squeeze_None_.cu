
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_40(float* __restrict__ conv2d_nchw, float* __restrict__ conv2d_nchw_1, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int ry = 0; ry < 3; ++ry) {
    for (int rx = 0; rx < 3; ++rx) {
      if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
        conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv2d_nchw_1[(((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 15) * 122) + (ry * 122)) + rx) + (((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 120))] * ph[((ry * 3) + rx)]));
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_10(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int ry = 0; ry < 3; ++ry) {
    for (int rx = 0; rx < 3; ++rx) {
      if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
        conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[(((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 15) * 122) + (ry * 122)) + rx) + (((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 120))] * ph[((ry * 3) + rx)]));
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_39(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_69(float* __restrict__ conv2d_nchw, float* __restrict__ conv2d_nchw_1, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((3721 <= ((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2))) ? conv2d_nchw_1[(((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3721) * 14400) + (((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 7442) / 61) * 120)) + (((((int)blockIdx.x) * 48) + ((int)threadIdx.x)) % 122)) - 14400)] : pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_66(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_32(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 5)) < 961) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (((((31 <= (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) % 3844)) && ((((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) % 15376) < 15252)) && (1 <= (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 124))) && ((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 124) < 123)) ? conv2d_nchw[(((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) / 961) * 14884) + (((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) % 3844) / 31) * 122)) + (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 124)) - 123)] : -3.402823e+38f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_26(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * ph[0]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ conv2d_nchw) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_13(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_45(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_11(float* __restrict__ conv2d_nchw) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_25(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_23(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 2; ++rc) {
    if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
      pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv2d_nchw[(((rc * 14884) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] * ph[rc]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_34(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 2; ++rc) {
    if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
      pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv2d_nchw[(((rc * 14884) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] * ph[rc]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_63(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_18(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 3; ++rc) {
    for (int ry = 0; ry < 7; ++ry) {
      for (int rx = 0; rx < 7; ++rx) {
        if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
          conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[(((((rc * 16384) + ((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) / 61) * 128)) + (ry * 128)) + rx) + (((((int)blockIdx.x) * 48) + ((int)threadIdx.x)) % 122))] * ph[(((rc * 49) + (ry * 7)) + rx)]));
        }
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ pad_temp, float* __restrict__ ph) {
  pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_75(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * ph[0]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_31(float* __restrict__ conv2d_nchw, float* __restrict__ conv2d_nchw_1, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((3721 <= ((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2))) ? conv2d_nchw_1[(((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3721) * 14400) + (((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 7442) / 61) * 120)) + (((((int)blockIdx.x) * 48) + ((int)threadIdx.x)) % 122)) - 14400)] : pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_36(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_21(float* __restrict__ conv2d_nchw) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_41(float* __restrict__ conv2d_nchw) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_78(float* __restrict__ conv2d_nchw, float* __restrict__ conv2d_nchw_1, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int ry = 0; ry < 3; ++ry) {
    for (int rx = 0; rx < 3; ++rx) {
      if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
        conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv2d_nchw_1[(((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 15) * 122) + (ry * 122)) + rx) + (((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 120))] * ph[((ry * 3) + rx)]));
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_55(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * ph[0]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_43(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 2; ++rc) {
    if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
      pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv2d_nchw[(((rc * 14884) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] * ph[rc]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_70(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 5)) < 961) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (((((31 <= (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) % 3844)) && ((((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) % 15376) < 15252)) && (1 <= (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 124))) && ((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 124) < 123)) ? conv2d_nchw[(((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) / 961) * 14884) + (((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) % 3844) / 31) * 122)) + (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 124)) - 123)] : -3.402823e+38f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_9(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_24(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_20(float* __restrict__ conv2d_nchw, float* __restrict__ conv2d_nchw_1, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int ry = 0; ry < 3; ++ry) {
    for (int rx = 0; rx < 3; ++rx) {
      if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
        conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv2d_nchw_1[(((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 15) * 122) + (ry * 122)) + rx) + (((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 120))] * ph[((ry * 3) + rx)]));
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_8(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_52(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 2; ++rc) {
    if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
      pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv2d_nchw[(((rc * 14884) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] * ph[rc]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_64(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * ph[0]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_15(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_48(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_59(float* __restrict__ conv2d_nchw) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_53(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_61(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 2; ++rc) {
    if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
      pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv2d_nchw[(((rc * 14884) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] * ph[rc]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_72(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 2; ++rc) {
    if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
      pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv2d_nchw[(((rc * 14884) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] * ph[rc]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_62(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_49(float* __restrict__ conv2d_nchw, float* __restrict__ conv2d_nchw_1, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int ry = 0; ry < 3; ++ry) {
    for (int rx = 0; rx < 3; ++rx) {
      if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
        conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv2d_nchw_1[(((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 15) * 122) + (ry * 122)) + rx) + (((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 120))] * ph[((ry * 3) + rx)]));
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_27(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_28(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_30(float* __restrict__ conv2d_nchw) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_22(float* __restrict__ conv2d_nchw, float* __restrict__ conv2d_nchw_1, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((3721 <= ((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2))) ? conv2d_nchw_1[(((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3721) * 14400) + (((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 7442) / 61) * 120)) + (((((int)blockIdx.x) * 48) + ((int)threadIdx.x)) % 122)) - 14400)] : pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_37(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * ph[0]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_5(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * ph[0]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_68(float* __restrict__ conv2d_nchw) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_14(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int rc = 0; rc < 2; ++rc) {
    if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
      pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv2d_nchw[(((rc * 14884) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] * ph[rc]));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_50(float* __restrict__ conv2d_nchw) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_77(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_79(float* __restrict__ conv2d_nchw) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_17(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * ph[0]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_54(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_42(float* __restrict__ conv2d_nchw, float* __restrict__ conv2d_nchw_1, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((3721 <= ((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2))) ? conv2d_nchw_1[(((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3721) * 14400) + (((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 7442) / 61) * 120)) + (((((int)blockIdx.x) * 48) + ((int)threadIdx.x)) % 122)) - 14400)] : pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 961) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (((((31 <= ((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2))) && (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3813)) && (1 <= (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 124))) && ((((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 124) < 123)) ? conv2d_nchw[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 31) * 122) + (((((int)blockIdx.x) * 32) + ((int)threadIdx.x)) % 124)) - 123)] : -3.402823e+38f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_16(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_56(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_60(float* __restrict__ conv2d_nchw, float* __restrict__ conv2d_nchw_1, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((3721 <= ((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2))) ? conv2d_nchw_1[(((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3721) * 14400) + (((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 7442) / 61) * 120)) + (((((int)blockIdx.x) * 48) + ((int)threadIdx.x)) % 122)) - 14400)] : pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_29(float* __restrict__ conv2d_nchw, float* __restrict__ conv2d_nchw_1, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int ry = 0; ry < 3; ++ry) {
    for (int rx = 0; rx < 3; ++rx) {
      if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
        conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv2d_nchw_1[(((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 15) * 122) + (ry * 122)) + rx) + (((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 120))] * ph[((ry * 3) + rx)]));
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_76(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_65(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_80(float* __restrict__ T_concat, float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 3721) {
    T_concat[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((3721 <= ((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2))) ? conv2d_nchw[(((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3721) * 14400) + (((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 7442) / 61) * 120)) + (((((int)blockIdx.x) * 48) + ((int)threadIdx.x)) % 122)) - 14400)] : pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_46(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * ph[0]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_47(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_6(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_71(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 3721) {
        conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], pad_temp[((((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3721) * 15376) + (((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 7442) / 61) * 124)) + (rv0 * 124)) + rv1) + (((((int)blockIdx.x) * 48) + ((int)threadIdx.x)) % 122))]);
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_19(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_51(float* __restrict__ conv2d_nchw, float* __restrict__ conv2d_nchw_1, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((3721 <= ((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2))) ? conv2d_nchw_1[(((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3721) * 14400) + (((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 7442) / 61) * 120)) + (((((int)blockIdx.x) * 48) + ((int)threadIdx.x)) % 122)) - 14400)] : pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_74(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_7(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * ph[0]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_33(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 3721) {
        conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], pad_temp[((((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3721) * 15376) + (((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 7442) / 61) * 124)) + (rv0 * 124)) + rv1) + (((((int)blockIdx.x) * 48) + ((int)threadIdx.x)) % 122))]);
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_44(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_12(float* __restrict__ conv2d_nchw, float* __restrict__ conv2d_nchw_1, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((3721 <= ((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2))) ? conv2d_nchw[(((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3721) * 14400) + (((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 7442) / 61) * 120)) + (((((int)blockIdx.x) * 48) + ((int)threadIdx.x)) % 122)) - 14400)] : conv2d_nchw_1[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_58(float* __restrict__ conv2d_nchw, float* __restrict__ conv2d_nchw_1, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int ry = 0; ry < 3; ++ry) {
    for (int rx = 0; rx < 3; ++rx) {
      if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
        conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv2d_nchw_1[(((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 15) * 122) + (ry * 122)) + rx) + (((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 120))] * ph[((ry * 3) + rx)]));
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_38(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_73(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_57(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_4(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
        conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], pad_temp[(((((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) / 61) * 124) + (rv0 * 124)) + rv1) + (((((int)blockIdx.x) * 48) + ((int)threadIdx.x)) % 122))]);
      }
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_35(float* __restrict__ conv2d_nchw, float* __restrict__ pad_temp) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3721) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(pad_temp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_67(float* __restrict__ conv2d_nchw, float* __restrict__ conv2d_nchw_1, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
    conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int ry = 0; ry < 3; ++ry) {
    for (int rx = 0; rx < 3; ++rx) {
      if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 225) {
        conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (conv2d_nchw[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + (conv2d_nchw_1[(((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 15) * 122) + (ry * 122)) + rx) + (((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 120))] * ph[((ry * 3) + rx)]));
      }
    }
  }
}

