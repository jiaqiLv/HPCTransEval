
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(18) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))] = atanf(data[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))]);
}

