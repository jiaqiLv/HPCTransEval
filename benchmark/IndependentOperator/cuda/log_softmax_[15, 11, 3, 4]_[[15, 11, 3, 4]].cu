
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(2) default_function_kernel_1(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) < 495) {
    compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k = 0; k < 4; ++k) {
    if (((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) < 495) {
      compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] = (compute[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))] + __expf((data[(((((int)blockIdx.x) * 8) + (((int)threadIdx.x) * 4)) + k)] - T_softmax_maxelem[((((int)blockIdx.x) * 2) + ((int)threadIdx.x))])));
    }
  }
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ T_softmax_maxelem, float* __restrict__ compute, float* __restrict__ compute_1, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2)) < 495) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ((data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - T_softmax_maxelem[((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2))]) - __logf(compute_1[((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 2))]));
  }
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) < 495) {
    T_softmax_maxelem[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int k = 0; k < 4; ++k) {
    if (((((int)blockIdx.x) * 8) + ((int)threadIdx.x)) < 495) {
      T_softmax_maxelem[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))], data[(((((int)blockIdx.x) * 32) + (((int)threadIdx.x) * 4)) + k)]);
    }
  }
}

