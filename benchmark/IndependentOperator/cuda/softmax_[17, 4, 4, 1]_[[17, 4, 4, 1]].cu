
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 17) {
    T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 4)) < 17) {
    T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))], data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_2(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_norm, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 4)) < 17) {
    T_softmax_norm[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (__expf((data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] - T_softmax_maxelem[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))])) / T_softmax_expsum[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_1(float* __restrict__ T_softmax_expsum, float* __restrict__ T_softmax_maxelem, float* __restrict__ data) {
  T_softmax_expsum[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = 0.000000e+00f;
  T_softmax_expsum[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = (T_softmax_expsum[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] + __expf((data[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] - T_softmax_maxelem[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))])));
}

