
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ data, float* __restrict__ pool_max) {
  pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      for (int rv2 = 0; rv2 < 3; ++rv2) {
        pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = max(pool_max[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))], (((((1 <= ((((((int)blockIdx.x) & 1) * 4) + ((((int)threadIdx.x) >> 5) * 2)) + rv0)) && (((((((int)threadIdx.x) >> 5) + (rv0 >> 1)) >> 1) + (((int)blockIdx.x) & 1)) < 2)) && (1 <= ((((((int)threadIdx.x) & 31) >> 3) * 2) + rv1))) && (1 <= (((((int)threadIdx.x) & 7) * 2) + rv2))) ? data[((((((((((((int)blockIdx.x) >> 1) * 896) + ((((int)blockIdx.x) & 1) * 512)) + ((((int)threadIdx.x) >> 5) * 256)) + (rv0 * 128)) + (((((int)threadIdx.x) & 31) >> 3) * 32)) + (rv1 * 16)) + ((((int)threadIdx.x) & 7) * 2)) + rv2) - 145)] : -3.402823e+38f));
      }
    }
  }
}

