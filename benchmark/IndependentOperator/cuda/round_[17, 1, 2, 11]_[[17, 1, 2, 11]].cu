
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(3) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 3) + ((int)threadIdx.x)) < 374) {
    compute[((((int)blockIdx.x) * 3) + ((int)threadIdx.x))] = roundf(data[((((int)blockIdx.x) * 3) + ((int)threadIdx.x))]);
  }
}

