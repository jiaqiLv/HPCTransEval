
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(26) default_function_kernel_1(float* __restrict__ T_divide, float* __restrict__ data, float* __restrict__ tensor) {
  T_divide[((((int)blockIdx.x) * 26) + ((int)threadIdx.x))] = (data[((((int)blockIdx.x) * 26) + ((int)threadIdx.x))] / powf((2.000000e+00f + (1.000000e-04f * tensor[((((int)blockIdx.x) * 26) + ((int)threadIdx.x))])), 7.500000e-01f));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ data, float* __restrict__ tensor) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 1755) {
    tensor[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) < 1755) {
    tensor[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (tensor[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] + (data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] * data[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
  }
}

