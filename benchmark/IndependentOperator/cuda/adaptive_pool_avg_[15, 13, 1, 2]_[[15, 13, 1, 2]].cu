
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(48) default_function_kernel_1(float* __restrict__ adaptive_pool_avg, float* __restrict__ adaptive_pool_sum) {
  adaptive_pool_avg[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] / (((float)(((((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 3)) & 7) + 1) % 8) == 0) ? (((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 3)) & 7) + 1) >> 3) : ((((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 3)) & 7) + 1) >> 3) + 1))) * ((float)(((((((((int)threadIdx.x) & 7) * 2) + 2) % 8) == 0) ? (((((int)threadIdx.x) & 7) + 1) >> 2) : ((((((int)threadIdx.x) & 7) + 1) >> 2) + 1)) - ((((int)threadIdx.x) & 7) >> 2)))));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ adaptive_pool_sum, float* __restrict__ data) {
  adaptive_pool_sum[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < (((((((((int)blockIdx.x) & 1) * 4) + (((int)threadIdx.x) >> 3)) + 1) % 8) == 0) ? ((((((int)threadIdx.x) + 8) >> 5) + (((int)blockIdx.x) & 1)) >> 1) : (((((((int)threadIdx.x) + 8) >> 5) + (((int)blockIdx.x) & 1)) >> 1) + 1)); ++rv0) {
    for (int rv1 = 0; rv1 < (((((((((int)threadIdx.x) & 7) * 2) + 2) % 8) == 0) ? (((((int)threadIdx.x) & 7) + 1) >> 2) : ((((((int)threadIdx.x) & 7) + 1) >> 2) + 1)) - ((((int)threadIdx.x) & 7) >> 2)); ++rv1) {
      adaptive_pool_sum[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = (adaptive_pool_sum[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] + data[(((((((int)blockIdx.x) >> 1) * 2) + (rv0 * 2)) + ((((int)threadIdx.x) & 7) >> 2)) + rv1)]);
    }
  }
}

