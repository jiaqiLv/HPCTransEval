
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 187) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = ceilf(data[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
  }
}

