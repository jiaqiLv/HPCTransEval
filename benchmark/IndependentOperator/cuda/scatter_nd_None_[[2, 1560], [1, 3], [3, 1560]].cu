
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ atomic_add_return, int* __restrict__ indices, float* __restrict__ scatter_nd_cuda, float* __restrict__ updates) {
  if (((((int)blockIdx.y) * 128) + (((int)threadIdx.x) >> 3)) < 195) {
    atomic_add_return[0] = atomicAdd((&(scatter_nd_cuda[(((indices[((int)blockIdx.x)] * 1560) + (((int)blockIdx.y) * 1024)) + ((int)threadIdx.x))])), updates[(((((int)blockIdx.x) * 1560) + (((int)blockIdx.y) * 1024)) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ data, float* __restrict__ scatter_nd_cuda) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 195) {
    scatter_nd_cuda[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = data[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

