
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(55) default_function_kernel(float* __restrict__ compute, float* __restrict__ data) {
  compute[((((int)blockIdx.x) * 55) + ((int)threadIdx.x))] = tanf(data[((((int)blockIdx.x) * 55) + ((int)threadIdx.x))]);
}

