
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel(float* __restrict__ A, float* __restrict__ W, float* __restrict__ group_conv2d_nchw) {
  float group_conv2d_nchw_local[1];
  __shared__ float pad_temp_shared[1];
  __shared__ float W_shared[1];
  group_conv2d_nchw_local[0] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
      for (int rx_outer = 0; rx_outer < 3; ++rx_outer) {
        __syncthreads();
        pad_temp_shared[0] = A[(((((((((int)blockIdx.z) * 2097152) + ((((int)blockIdx.y) >> 4) * 524288)) + (rc_outer * 65536)) + ((((int)blockIdx.x) / 254) * 256)) + (ry_outer * 256)) + rx_outer) + (((int)blockIdx.x) % 254))];
        W_shared[0] = W[((((((int)blockIdx.y) * 72) + (rc_outer * 9)) + (ry_outer * 3)) + rx_outer)];
        __syncthreads();
        group_conv2d_nchw_local[0] = (group_conv2d_nchw_local[0] + (pad_temp_shared[0] * W_shared[0]));
      }
    }
  }
  group_conv2d_nchw[(((((int)blockIdx.z) * 4129024) + (((int)blockIdx.y) * 64516)) + ((int)blockIdx.x))] = group_conv2d_nchw_local[0];
}

