
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ Im, float* __restrict__ Re, float* __restrict__ dft_cuda, float* __restrict__ dft_cuda_1) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 75) {
    for (int i = 0; i < 50; ++i) {
      dft_cuda[(((((int)blockIdx.x) * 51200) + (((int)threadIdx.x) * 50)) + i)] = 0.000000e+00f;
      dft_cuda_1[(((((int)blockIdx.x) * 51200) + (((int)threadIdx.x) * 50)) + i)] = 0.000000e+00f;
      for (int j = 0; j < 50; ++j) {
        float cse_var_1 = (((-6.283185e+00f * ((float)i)) * 2.000000e-02f) * ((float)j));
        dft_cuda[(((((int)blockIdx.x) * 51200) + (((int)threadIdx.x) * 50)) + i)] = (dft_cuda[(((((int)blockIdx.x) * 51200) + (((int)threadIdx.x) * 50)) + i)] + ((Re[(((((int)blockIdx.x) * 51200) + (((int)threadIdx.x) * 50)) + j)] * __cosf(cse_var_1)) - (Im[(((((int)blockIdx.x) * 51200) + (((int)threadIdx.x) * 50)) + j)] * __sinf(cse_var_1))));
        dft_cuda_1[(((((int)blockIdx.x) * 51200) + (((int)threadIdx.x) * 50)) + i)] = (dft_cuda_1[(((((int)blockIdx.x) * 51200) + (((int)threadIdx.x) * 50)) + i)] + ((Re[(((((int)blockIdx.x) * 51200) + (((int)threadIdx.x) * 50)) + j)] * __sinf(cse_var_1)) + (Im[(((((int)blockIdx.x) * 51200) + (((int)threadIdx.x) * 50)) + j)] * __cosf(cse_var_1))));
      }
    }
  }
}

