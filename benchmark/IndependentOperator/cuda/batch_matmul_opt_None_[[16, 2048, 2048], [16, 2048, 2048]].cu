
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ T_batch_matmul_NT, float* __restrict__ x, float* __restrict__ y) {
  float T_batch_matmul_NT_local[64];
  __shared__ float x_shared[512];
  __shared__ float y_shared[512];
  float x_shared_local[8];
  float y_shared_local[8];
  for (int i_c_init = 0; i_c_init < 8; ++i_c_init) {
    for (int j_c_init = 0; j_c_init < 8; ++j_c_init) {
      T_batch_matmul_NT_local[((i_c_init * 8) + j_c_init)] = 0.000000e+00f;
    }
  }
  for (int k_outer = 0; k_outer < 256; ++k_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax1_inner = 0; ax1_inner < 8; ++ax1_inner) {
      x_shared[(((((int)threadIdx.y) * 64) + (ax1_inner * 8)) + ((int)threadIdx.x))] = x[((((((((int)blockIdx.z) * 4194304) + (((int)blockIdx.y) * 131072)) + (((int)threadIdx.y) * 16384)) + (ax1_inner * 2048)) + (k_outer * 8)) + ((int)threadIdx.x))];
    }
    #pragma unroll
    for (int ax1_inner_1 = 0; ax1_inner_1 < 8; ++ax1_inner_1) {
      y_shared[(((((int)threadIdx.y) * 64) + (ax1_inner_1 * 8)) + ((int)threadIdx.x))] = y[((((((((int)blockIdx.z) * 4194304) + (((int)blockIdx.x) * 131072)) + (((int)threadIdx.y) * 16384)) + (ax1_inner_1 * 2048)) + (k_outer * 8)) + ((int)threadIdx.x))];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 8; ++k_inner) {
      #pragma unroll
      for (int ax1 = 0; ax1 < 8; ++ax1) {
        x_shared_local[ax1] = x_shared[(((((int)threadIdx.y) * 64) + (ax1 * 8)) + k_inner)];
      }
      #pragma unroll
      for (int ax1_1 = 0; ax1_1 < 8; ++ax1_1) {
        y_shared_local[ax1_1] = y_shared[(((((int)threadIdx.x) * 64) + (ax1_1 * 8)) + k_inner)];
      }
      for (int i_c = 0; i_c < 8; ++i_c) {
        #pragma unroll
        for (int j_c = 0; j_c < 8; ++j_c) {
          T_batch_matmul_NT_local[((i_c * 8) + j_c)] = (T_batch_matmul_NT_local[((i_c * 8) + j_c)] + (x_shared_local[i_c] * y_shared_local[j_c]));
        }
      }
    }
  }
  for (int i_inner_inner = 0; i_inner_inner < 8; ++i_inner_inner) {
    #pragma unroll
    for (int j_inner_inner = 0; j_inner_inner < 8; ++j_inner_inner) {
      T_batch_matmul_NT[(((((((((int)blockIdx.z) * 4194304) + (((int)blockIdx.y) * 131072)) + (((int)threadIdx.y) * 16384)) + (i_inner_inner * 2048)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 8)) + j_inner_inner)] = T_batch_matmul_NT_local[((i_inner_inner * 8) + j_inner_inner)];
    }
  }
}

