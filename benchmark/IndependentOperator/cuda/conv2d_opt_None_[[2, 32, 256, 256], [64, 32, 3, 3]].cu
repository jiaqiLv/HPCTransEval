
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ conv2d_nchw, float* __restrict__ data, float* __restrict__ kernel) {
  float conv2d_nchw_local[8];
  __shared__ float pad_temp_shared[32];
  __shared__ float kernel_shared[576];
  for (int yy_c_init = 0; yy_c_init < 2; ++yy_c_init) {
    conv2d_nchw_local[yy_c_init] = 0.000000e+00f;
    conv2d_nchw_local[(yy_c_init + 2)] = 0.000000e+00f;
    conv2d_nchw_local[(yy_c_init + 4)] = 0.000000e+00f;
    conv2d_nchw_local[(yy_c_init + 6)] = 0.000000e+00f;
  }
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      pad_temp_shared[(((((int)threadIdx.z) * 4) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)] = data[(((((((((((int)blockIdx.z) >> 1) * 2097152) + (rc_outer * 131072)) + ((((int)threadIdx.z) >> 2) * 65536)) + (((int)blockIdx.y) * 512)) + ((((int)threadIdx.z) & 3) * 256)) + (((int)blockIdx.x) * 2)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner)];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1 < 36; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1) {
      kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 36)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1)] = kernel[(((((((((int)blockIdx.z) & 1) * 9216) + (((int)threadIdx.z) * 1152)) + (((int)threadIdx.x) * 576)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1 / 18) * 288)) + (rc_outer * 18)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner_1 % 18))];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
      for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
        for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
          for (int yy_c = 0; yy_c < 2; ++yy_c) {
            conv2d_nchw_local[yy_c] = (conv2d_nchw_local[yy_c] + (pad_temp_shared[(((((rc_inner * 16) + (yy_c * 4)) + (ry_inner * 4)) + ((int)threadIdx.x)) + rx_inner)] * kernel_shared[((((((int)threadIdx.z) * 18) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner)]));
            conv2d_nchw_local[(yy_c + 2)] = (conv2d_nchw_local[(yy_c + 2)] + (pad_temp_shared[(((((rc_inner * 16) + (yy_c * 4)) + (ry_inner * 4)) + ((int)threadIdx.x)) + rx_inner)] * kernel_shared[(((((((int)threadIdx.z) * 18) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 144)]));
            conv2d_nchw_local[(yy_c + 4)] = (conv2d_nchw_local[(yy_c + 4)] + (pad_temp_shared[(((((rc_inner * 16) + (yy_c * 4)) + (ry_inner * 4)) + ((int)threadIdx.x)) + rx_inner)] * kernel_shared[(((((((int)threadIdx.z) * 18) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 288)]));
            conv2d_nchw_local[(yy_c + 6)] = (conv2d_nchw_local[(yy_c + 6)] + (pad_temp_shared[(((((rc_inner * 16) + (yy_c * 4)) + (ry_inner * 4)) + ((int)threadIdx.x)) + rx_inner)] * kernel_shared[(((((((int)threadIdx.z) * 18) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner) + 432)]));
          }
        }
      }
    }
  }
  for (int yy_inner_inner_inner = 0; yy_inner_inner_inner < 2; ++yy_inner_inner_inner) {
    conv2d_nchw[((((((((int)blockIdx.z) * 2064512) + (((int)threadIdx.z) * 64516)) + (((int)blockIdx.y) * 508)) + (yy_inner_inner_inner * 254)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x))] = conv2d_nchw_local[yy_inner_inner_inner];
    conv2d_nchw[(((((((((int)blockIdx.z) * 2064512) + (((int)threadIdx.z) * 64516)) + (((int)blockIdx.y) * 508)) + (yy_inner_inner_inner * 254)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 516128)] = conv2d_nchw_local[(yy_inner_inner_inner + 2)];
    conv2d_nchw[(((((((((int)blockIdx.z) * 2064512) + (((int)threadIdx.z) * 64516)) + (((int)blockIdx.y) * 508)) + (yy_inner_inner_inner * 254)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 1032256)] = conv2d_nchw_local[(yy_inner_inner_inner + 4)];
    conv2d_nchw[(((((((((int)blockIdx.z) * 2064512) + (((int)threadIdx.z) * 64516)) + (((int)blockIdx.y) * 508)) + (yy_inner_inner_inner * 254)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 1548384)] = conv2d_nchw_local[(yy_inner_inner_inner + 6)];
  }
}

