
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(100) default_function_kernel_8(float* __restrict__ T_cast, float* __restrict__ ph) {
  T_cast[((int)threadIdx.x)] = ph[((int)threadIdx.x)];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_9(float* __restrict__ T_concat, float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 1125) {
    T_concat[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(180) default_function_kernel_10(float* __restrict__ T_concat, float* __restrict__ T_multiply_red) {
  T_multiply_red[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k1 = 0; k1 < 10; ++k1) {
    for (int k2 = 0; k2 < 10; ++k2) {
      T_multiply_red[((int)threadIdx.x)] = (T_multiply_red[((int)threadIdx.x)] + T_concat[(((((((int)threadIdx.x) / 9) * 900) + (k1 * 90)) + (k2 * 9)) + (((int)threadIdx.x) % 9))]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_12(float* __restrict__ T_cast, float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 1125) {
    T_cast[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_concat, float* __restrict__ T_strided_slice, float* __restrict__ T_strided_slice_1) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 1125) {
    T_concat[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((1125 <= ((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3))) ? T_strided_slice[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) - 9000)] : T_strided_slice_1[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_6(float* __restrict__ T_concat, float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 1125) {
    T_concat[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_concat[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] / T_strided_slice[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) / 9)]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_strided_slice, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 1125) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 225) * 31944) + (((((((int)blockIdx.x) * 62) + (((int)threadIdx.x) >> 1)) % 450) / 45) * 1452)) + (((((((int)blockIdx.x) * 34) + ((int)threadIdx.x)) % 90) / 9) * 44)) + (((((int)blockIdx.x) * 7) + ((int)threadIdx.x)) % 9)) + 16973)];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_7(float* __restrict__ T_concat, float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 1125) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_concat[(((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 225) * 900) + (((((((int)blockIdx.x) * 34) + ((int)threadIdx.x)) % 90) / 9) * 90)) + (((((((int)blockIdx.x) * 62) + (((int)threadIdx.x) >> 1)) % 450) / 45) * 9)) + (((((int)blockIdx.x) * 7) + ((int)threadIdx.x)) % 9))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_5(float* __restrict__ T_concat, float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 125) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k = 0; k < 9; ++k) {
    if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 125) {
      T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + T_concat[(((((int)blockIdx.x) * 9216) + (((int)threadIdx.x) * 9)) + k)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_strided_slice, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 1125) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 225) * 31944) + (((((((int)blockIdx.x) * 62) + (((int)threadIdx.x) >> 1)) % 450) / 45) * 1452)) + (((((((int)blockIdx.x) * 34) + ((int)threadIdx.x)) % 90) / 9) * 44)) + (((((int)blockIdx.x) * 7) + ((int)threadIdx.x)) % 9)) + 50414)];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_11(float* __restrict__ T_cast, float* __restrict__ T_multiply_red, float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 1125) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_cast[((((((((int)blockIdx.x) * 62) + (((int)threadIdx.x) >> 1)) % 450) / 45) * 10) + ((((((int)blockIdx.x) * 34) + ((int)threadIdx.x)) % 90) / 9))]) * (1.000000e+00f / sqrtf(((T_multiply_red[(((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 225) * 9) + (((((int)blockIdx.x) * 7) + ((int)threadIdx.x)) % 9))] * 1.000000e-02f) + 1.000000e-05f))));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ T_concat, float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 125) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int k = 0; k < 9; ++k) {
    if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 125) {
      T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], T_concat[(((((int)blockIdx.x) * 9216) + (((int)threadIdx.x) * 9)) + k)]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_4(float* __restrict__ T_concat, float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 1125) {
    T_concat[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = __expf((T_concat[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] - T_strided_slice[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) / 9)]));
  }
}

