
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel_2(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  T_mod[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = fmodf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))], ceilf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]));
}

extern "C" __global__ void default_function_kernel_1(float* __restrict__ T_multiply, float* __restrict__ ph_0) {
  T_multiply[((int)blockIdx.x)] = (atanf(ph_0[((int)blockIdx.x)]) * ph_0[((int)blockIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = acosf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}

