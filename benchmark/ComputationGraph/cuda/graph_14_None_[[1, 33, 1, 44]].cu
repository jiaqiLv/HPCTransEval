
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_broadcast_to) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 11979) {
    T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = sqrtf(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_broadcast_to, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3993) {
    T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) % 1452)];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_4(float* __restrict__ T_broadcast_to, float* __restrict__ T_broadcast_to_1) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 11979) {
    T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] - T_broadcast_to_1[(((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) % 1089) / 33) * 528) + (((((((int)blockIdx.x) * 124) + (((int)threadIdx.x) >> 2)) % 132) / 11) * 44)) + (((((int)blockIdx.x) * 12) + ((int)threadIdx.x)) % 44))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_6(float* __restrict__ T_broadcast_to, float* __restrict__ T_broadcast_to_1) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 1089) {
    T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k = 0; k < 11; ++k) {
    if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 1089) {
      T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + T_broadcast_to_1[(((k * 17424) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_broadcast_to, float* __restrict__ T_broadcast_to_1) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 11979) {
    T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_broadcast_to_1[(((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) / 33) * 44) + (((((int)blockIdx.x) * 12) + ((int)threadIdx.x)) % 44))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_5(float* __restrict__ T_broadcast_to) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 11979) {
      int v_ = ((int)(floorf(((max(min(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
    T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_7(float* __restrict__ T_broadcast_to, float* __restrict__ T_broadcast_to_1, float* __restrict__ T_softmax_norm) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 11979) {
    T_softmax_norm[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] / T_broadcast_to_1[(((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) % 1089) / 33) * 528) + (((((((int)blockIdx.x) * 124) + (((int)threadIdx.x) >> 2)) % 132) / 11) * 44)) + (((((int)blockIdx.x) * 12) + ((int)threadIdx.x)) % 44))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ T_broadcast_to, float* __restrict__ T_broadcast_to_1) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 1089) {
    T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int k = 0; k < 11; ++k) {
    if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 1089) {
      T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], T_broadcast_to_1[(((k * 17424) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))]);
    }
  }
}

