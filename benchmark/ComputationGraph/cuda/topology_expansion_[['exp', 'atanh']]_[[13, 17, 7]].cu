
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = atanhf(ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(9) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))] = __expf(ph_0[((((int)blockIdx.x) * 9) + ((int)threadIdx.x))]);
}

