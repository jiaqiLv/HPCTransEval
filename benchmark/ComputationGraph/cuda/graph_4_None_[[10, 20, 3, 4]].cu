
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_reshape) {
  if (((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 5)) < 45) {
      int v_ = ((int)(floorf(((max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
    T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ T_reshape, float* __restrict__ T_reshape_1) {
  if (((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 5)) < 45) {
    T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_reshape_1[((((((((((int)blockIdx.x) * 40) + (((int)threadIdx.x) >> 2)) % 72) / 3) * 60) + ((((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 5)) / 9) * 12)) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 6) * 2)) + ((((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 1)) % 6) / 3))];
  }
}

extern "C" __global__ void __launch_bounds__(60) default_function_kernel_4(float* __restrict__ T_reshape, float* __restrict__ T_transpose_red) {
  T_transpose_red[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k1 = 0; k1 < 24; ++k1) {
    T_transpose_red[((int)threadIdx.x)] = (T_transpose_red[((int)threadIdx.x)] + T_reshape[((((((int)threadIdx.x) / 12) * 288) + (k1 * 12)) + (((int)threadIdx.x) % 12))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_reshape, float* __restrict__ T_reshape_1) {
  if (((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 5)) < 45) {
    T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_reshape_1[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 15) * 60) + ((((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 1)) % 30) * 2)) + (((int)threadIdx.x) & 1))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_reshape, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 32) + (((int)threadIdx.x) >> 5)) < 45) {
    T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[(((((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) / 15) * 240) + (((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 2)) % 60) / 3) * 12)) + ((((((int)threadIdx.x) >> 2) + ((int)blockIdx.x)) % 3) * 4)) + (((int)threadIdx.x) & 3))];
  }
}

