
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(840) default_function_kernel_3(float* __restrict__ T_reshape, float* __restrict__ T_transpose) {
  T_transpose[((int)threadIdx.x)] = T_reshape[(((((((int)threadIdx.x) % 28) / 7) * 210) + ((((int)threadIdx.x) / 28) * 7)) + (((int)threadIdx.x) % 7))];
}

extern "C" __global__ void __launch_bounds__(840) default_function_kernel_1(float* __restrict__ T_reshape) {
  T_reshape[((int)threadIdx.x)] = max(min(T_reshape[((int)threadIdx.x)], 9.999000e+01f), 1.111000e+01f);
}

extern "C" __global__ void __launch_bounds__(840) default_function_kernel_2(float* __restrict__ T_reshape) {
    int v_ = ((int)(floorf(((max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
  T_reshape[((int)threadIdx.x)] = max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), T_reshape[((int)threadIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(840) default_function_kernel(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = ph[((int)threadIdx.x)];
}

