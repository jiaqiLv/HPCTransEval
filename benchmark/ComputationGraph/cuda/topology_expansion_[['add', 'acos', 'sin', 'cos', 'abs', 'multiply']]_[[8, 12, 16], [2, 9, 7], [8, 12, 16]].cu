
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = __cosf(__sinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_3(float* __restrict__ T_multiply, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_multiply[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = ((ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] + ph_3[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]) * ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fabsf(__sinf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = acosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

