
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_strided_slice, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 125) * 79860) + (((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 2)) % 250) / 25) * 2420)) + (((((((int)blockIdx.x) * 12) + (((int)threadIdx.x) >> 1)) % 50) / 5) * 55)) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 10)) + 933559)];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_strided_slice, float* __restrict__ compute) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    compute[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = atanhf(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_strided_slice, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 125) * 31944) + (((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 2)) % 250) / 25) * 1452)) + (((((((int)blockIdx.x) * 12) + (((int)threadIdx.x) >> 1)) % 50) / 5) * 44)) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 10)) + 16973)];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ T_reverse_sequence, float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_strided_slice[(((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 100) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 10)) + 90) - (((((((int)blockIdx.x) * 12) + (((int)threadIdx.x) >> 1)) % 50) / 5) * 10))];
  }
}

