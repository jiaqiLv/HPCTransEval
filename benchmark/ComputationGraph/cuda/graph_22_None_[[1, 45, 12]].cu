
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_broadcast_to, float* __restrict__ T_transpose) {
  if (((int)threadIdx.x) < 75) {
    T_broadcast_to[((int)threadIdx.x)] = T_transpose[(((int)threadIdx.x) / 3)];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_broadcast_to) {
  if (((int)threadIdx.x) < 75) {
    T_broadcast_to[((int)threadIdx.x)] = floorf((T_broadcast_to[((int)threadIdx.x)] * 3.333333e-01f));
  }
}

extern "C" __global__ void __launch_bounds__(75) default_function_kernel_4(float* __restrict__ T_broadcast_to, float* __restrict__ T_strided_slice) {
  T_strided_slice[((int)threadIdx.x)] = T_broadcast_to[((int)threadIdx.x)];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ T_broadcast_to) {
  if (((int)threadIdx.x) < 75) {
    T_broadcast_to[((int)threadIdx.x)] = ((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * -2.726142e-10f) + 2.770681e-08f)) + -2.101024e-06f)) + -5.692506e-05f)) + -7.349906e-04f)) + -2.954600e-03f)) + -1.609603e-02f)) / (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * -1.456607e-05f) + -2.133740e-04f)) + -1.682827e-03f)) + -7.373329e-03f)) + -1.426474e-02f));
  }
}

extern "C" __global__ void __launch_bounds__(540) default_function_kernel(float* __restrict__ T_transpose, float* __restrict__ ph) {
  if (((int)threadIdx.x) < 25) {
    T_transpose[((int)threadIdx.x)] = ph[((((((int)threadIdx.x) / 5) * 12) + (((int)threadIdx.x) % 5)) + 14)];
  }
}

