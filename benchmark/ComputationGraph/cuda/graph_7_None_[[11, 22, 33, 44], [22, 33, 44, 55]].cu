
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_strided_slice, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 125) * 79860) + (((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 2)) % 250) / 25) * 2420)) + (((((((int)blockIdx.x) * 12) + (((int)threadIdx.x) >> 1)) % 50) / 5) * 55)) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 10)) + 933559)];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ T_fast_exp, float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
      int v_ = ((int)(floorf(((max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
    T_fast_exp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_strided_slice, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 125) * 31944) + (((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 2)) % 250) / 25) * 1452)) + (((((((int)blockIdx.x) * 12) + (((int)threadIdx.x) >> 1)) % 50) / 5) * 44)) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 10)) + 16973)];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_strided_slice, float* __restrict__ T_strided_slice_1) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_strided_slice_1[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

