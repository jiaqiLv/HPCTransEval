
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = atanf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(36) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))] = atanf(ph_0[((((int)blockIdx.x) * 36) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fabsf(atanf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

