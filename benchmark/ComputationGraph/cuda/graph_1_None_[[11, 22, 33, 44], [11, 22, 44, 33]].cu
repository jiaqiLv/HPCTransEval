
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_4(float* __restrict__ T_transpose, float* __restrict__ T_transpose_1) {
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 32768) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) >> 3)) < 43923) {
      T_transpose[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = T_transpose_1[(((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 32768) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) >> 3)) % 1331) / 121) * 31944) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 32768) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) >> 3)) / 1331) * 968)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x)) % 44) * 22)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) >> 2)) % 242) / 11))];
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_transpose, float* __restrict__ ph) {
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 32768) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) >> 3)) < 43923) {
      T_transpose[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = ph[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x)) % 11) * 31944) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x)) % 7986) / 363) * 1452)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 131072) + (((int)blockIdx.x) * 512)) + (((int)threadIdx.x) >> 1)) / 3993) * 33)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x)) % 363) / 11))];
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_transpose, float* __restrict__ T_transpose_1) {
  for (int ax0_ax1_fused_outer = 0; ax0_ax1_fused_outer < 2; ++ax0_ax1_fused_outer) {
    if ((((ax0_ax1_fused_outer * 32768) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) >> 3)) < 43923) {
      T_transpose[(((ax0_ax1_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = T_transpose_1[((((((ax0_ax1_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x)) % 968) * 363) + ((((ax0_ax1_fused_outer * 32768) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) >> 3)) / 121))];
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_transpose, float* __restrict__ ph) {
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 32768) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) >> 3)) < 43923) {
      T_transpose[(((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = ph[((((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 32768) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) >> 3)) / 3993) * 31944) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + (((int)threadIdx.x) >> 2)) % 242) / 11) * 1452)) + ((((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 32768) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) >> 3)) % 3993) / 121) * 44)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x)) % 44))];
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ T_transpose, float* __restrict__ T_transpose_1) {
  for (int ax0_ax1_fused_outer = 0; ax0_ax1_fused_outer < 2; ++ax0_ax1_fused_outer) {
    if ((((ax0_ax1_fused_outer * 32768) + (((int)blockIdx.x) * 128)) + (((int)threadIdx.x) >> 3)) < 43923) {
      T_transpose[(((ax0_ax1_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] = (T_transpose[(((ax0_ax1_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))] + T_transpose_1[(((ax0_ax1_fused_outer * 262144) + (((int)blockIdx.x) * 1024)) + ((int)threadIdx.x))]);
    }
  }
}

