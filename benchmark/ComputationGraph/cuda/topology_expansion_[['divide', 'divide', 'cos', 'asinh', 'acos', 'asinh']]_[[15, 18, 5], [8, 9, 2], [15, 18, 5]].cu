
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_2(float* __restrict__ T_divide, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_divide[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = asinhf(__cosf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(48) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))] = acosf(__cosf(ph_0[((((int)blockIdx.x) * 48) + ((int)threadIdx.x))]));
}

