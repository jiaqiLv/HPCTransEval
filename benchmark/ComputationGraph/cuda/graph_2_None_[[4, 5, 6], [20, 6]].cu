
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(120) default_function_kernel_7(float* __restrict__ T_reshape, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_norm) {
  T_softmax_norm[((int)threadIdx.x)] = (T_reshape[((int)threadIdx.x)] / T_softmax_maxelem[(((int)threadIdx.x) / 6)]);
}

extern "C" __global__ void __launch_bounds__(20) default_function_kernel_3(float* __restrict__ T_reshape, float* __restrict__ T_softmax_maxelem) {
  T_softmax_maxelem[((int)threadIdx.x)] = -3.402823e+38f;
  for (int k = 0; k < 6; ++k) {
    T_softmax_maxelem[((int)threadIdx.x)] = max(T_softmax_maxelem[((int)threadIdx.x)], T_reshape[((((int)threadIdx.x) * 6) + k)]);
  }
}

extern "C" __global__ void __launch_bounds__(120) default_function_kernel_2(float* __restrict__ T_reshape) {
  T_reshape[((int)threadIdx.x)] = max(T_reshape[((int)threadIdx.x)], 0.000000e+00f);
}

extern "C" __global__ void __launch_bounds__(120) default_function_kernel(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = ph[((int)threadIdx.x)];
}

extern "C" __global__ void __launch_bounds__(20) default_function_kernel_6(float* __restrict__ T_reshape, float* __restrict__ T_softmax_maxelem) {
  T_softmax_maxelem[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k = 0; k < 6; ++k) {
    T_softmax_maxelem[((int)threadIdx.x)] = (T_softmax_maxelem[((int)threadIdx.x)] + T_reshape[((((int)threadIdx.x) * 6) + k)]);
  }
}

extern "C" __global__ void __launch_bounds__(120) default_function_kernel_1(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((int)threadIdx.x)] = (T_reshape[((int)threadIdx.x)] - ph[((int)threadIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(120) default_function_kernel_4(float* __restrict__ T_reshape, float* __restrict__ T_softmax_maxelem) {
  T_reshape[((int)threadIdx.x)] = (T_reshape[((int)threadIdx.x)] - T_softmax_maxelem[(((int)threadIdx.x) / 6)]);
}

extern "C" __global__ void __launch_bounds__(120) default_function_kernel_5(float* __restrict__ T_reshape) {
    int v_ = ((int)(floorf(((max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
  T_reshape[((int)threadIdx.x)] = max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), T_reshape[((int)threadIdx.x)]);
}

