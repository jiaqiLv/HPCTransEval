
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_transpose, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 7)) < 15) {
    T_transpose[(((((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) / 3) * 192) + (((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) % 24) / 3) * 24)) + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) % 6) * 4)) + (((int)threadIdx.x) & 3))] = ph[((((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) % 6) * 14688) + (((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) % 24) / 3) * 272)) + ((((int)threadIdx.x) & 3) * 17)) + (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) / 3)) + 29699)];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_transpose) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 7)) < 15) {
    T_transpose[(((((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) / 3) * 192) + (((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) % 24) / 3) * 24)) + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) % 6) * 4)) + (((int)threadIdx.x) & 3))] = __log2f(T_transpose[(((((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) / 3) * 192) + (((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) % 24) / 3) * 24)) + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) % 6) * 4)) + (((int)threadIdx.x) & 3))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_4(float* __restrict__ T_fast_exp, float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 7)) < 15) {
      int v_ = ((int)(floorf(((max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
    T_fast_exp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_transpose) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 7)) < 15) {
    T_transpose[(((((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) / 3) * 192) + (((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) % 24) / 3) * 24)) + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) % 6) * 4)) + (((int)threadIdx.x) & 3))] = __cosf(T_transpose[(((((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) / 3) * 192) + (((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) % 24) / 3) * 24)) + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) % 6) * 4)) + (((int)threadIdx.x) & 3))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ T_strided_slice, float* __restrict__ T_transpose) {
  if (((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 7)) < 15) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_transpose[(((((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) / 3) * 192) + (((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 3)) % 24) / 3) * 24)) + ((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 2)) % 6) * 4)) + (((int)threadIdx.x) & 3))];
  }
}

