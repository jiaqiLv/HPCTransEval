
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ T_transpose) {
  T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (1.100000e+01f - T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_reshape, float* __restrict__ T_transpose) {
  T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_reshape[((((((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 120) * 128) + (((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) % 480) / 15) * 4)) + (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 8)) / 15))];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_reshape, float* __restrict__ ph) {
  T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))];
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_transpose) {
  T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + 3.000000e+00f);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_4(float* __restrict__ T_transpose, float* __restrict__ ph) {
  T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * ph[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_5(float* __restrict__ T_power, float* __restrict__ T_transpose) {
  T_power[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = powf(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 2.000000e+00f);
}

