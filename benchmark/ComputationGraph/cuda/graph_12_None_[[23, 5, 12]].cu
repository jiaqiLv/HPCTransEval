
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_5(float* __restrict__ T_broadcast_to) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 1035) {
    T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = min(9.000000e+00f, T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_7(float* __restrict__ T_broadcast_to, float* __restrict__ T_fast_tanh) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 1035) {
    T_fast_tanh[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * (((T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * -2.760768e-16f) + 2.000188e-13f)) + -8.604672e-11f)) + 5.122297e-08f)) + 1.485722e-05f)) + 6.372619e-04f)) + 4.893525e-03f)) / (((T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * 1.198258e-06f) + 1.185347e-04f)) + 2.268435e-03f)) + 4.893525e-03f));
  }
}

extern "C" __global__ void __launch_bounds__(69) default_function_kernel_3(float* __restrict__ T_broadcast_to, float* __restrict__ T_softmax_maxelem, float* __restrict__ compute) {
  compute[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k = 0; k < 60; ++k) {
    compute[((int)threadIdx.x)] = (compute[((int)threadIdx.x)] + __expf((T_broadcast_to[((((int)threadIdx.x) * 60) + k)] - T_softmax_maxelem[((int)threadIdx.x)])));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_reshape, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 345) {
    T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[(((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 15) * 60) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 60))];
  }
}

extern "C" __global__ void __launch_bounds__(69) default_function_kernel_2(float* __restrict__ T_broadcast_to, float* __restrict__ T_softmax_maxelem) {
  T_softmax_maxelem[((int)threadIdx.x)] = -3.402823e+38f;
  for (int k = 0; k < 60; ++k) {
    T_softmax_maxelem[((int)threadIdx.x)] = max(T_softmax_maxelem[((int)threadIdx.x)], T_broadcast_to[((((int)threadIdx.x) * 60) + k)]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_broadcast_to, float* __restrict__ T_reshape) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 1035) {
    T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_reshape[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) % 1380)];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_4(float* __restrict__ T_broadcast_to, float* __restrict__ T_softmax_maxelem, float* __restrict__ compute) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 1035) {
    T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 15)]) - __logf(compute[(((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 15)]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_6(float* __restrict__ T_broadcast_to) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 1035) {
    T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(-9.000000e+00f, T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

