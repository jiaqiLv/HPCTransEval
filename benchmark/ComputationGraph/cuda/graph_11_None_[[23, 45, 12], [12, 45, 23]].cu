
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_transpose, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3105) {
    T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((((((int)blockIdx.x) * 12) + ((int)threadIdx.x)) % 23) * 540) + (((((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) % 1035) / 23) * 12)) + (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) / 1035))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_transpose, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3105) {
    T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] / ph[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(540) default_function_kernel_5(float* __restrict__ T_softmax_maxelem, float* __restrict__ T_transpose) {
  T_softmax_maxelem[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k = 0; k < 23; ++k) {
    T_softmax_maxelem[((int)threadIdx.x)] = (T_softmax_maxelem[((int)threadIdx.x)] + T_transpose[((((int)threadIdx.x) * 23) + k)]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ T_softmax_maxelem, float* __restrict__ T_transpose) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3105) {
    T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) / 23)]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_4(float* __restrict__ T_transpose) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3105) {
      int v_ = ((int)(floorf(((max(min(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
    T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(540) default_function_kernel_2(float* __restrict__ T_softmax_maxelem, float* __restrict__ T_transpose) {
  T_softmax_maxelem[((int)threadIdx.x)] = -3.402823e+38f;
  for (int k = 0; k < 23; ++k) {
    T_softmax_maxelem[((int)threadIdx.x)] = max(T_softmax_maxelem[((int)threadIdx.x)], T_transpose[((((int)threadIdx.x) * 23) + k)]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_6(float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_norm, float* __restrict__ T_transpose) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3105) {
    T_softmax_norm[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] / T_softmax_maxelem[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) / 23)]);
  }
}

