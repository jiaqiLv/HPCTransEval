
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_reshape, float* __restrict__ T_softmax_maxelem) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 2541) {
    T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) / 21)]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_6(float* __restrict__ T_broadcast_to, float* __restrict__ T_reshape) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 17787) {
    T_broadcast_to[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_reshape[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 1617) * 924) + (((((((int)blockIdx.x) * 50) + (((int)threadIdx.x) >> 1)) % 462) / 21) * 42)) + (((((int)blockIdx.x) * 16) + ((int)threadIdx.x)) % 42))];
  }
}

extern "C" __global__ void __launch_bounds__(242) default_function_kernel_4(float* __restrict__ T_reshape, float* __restrict__ T_softmax_maxelem) {
  T_softmax_maxelem[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k = 0; k < 42; ++k) {
    T_softmax_maxelem[((int)threadIdx.x)] = (T_softmax_maxelem[((int)threadIdx.x)] + T_reshape[((((int)threadIdx.x) * 42) + k)]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_7(float* __restrict__ T_broadcast_to, float* __restrict__ T_reverse_sequence) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 17787) {
    T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_broadcast_to[(((((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) % 6468) + 64680) - ((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 1617) * 6468))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_5(float* __restrict__ T_reshape, float* __restrict__ T_softmax_maxelem) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 2541) {
    T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] / T_softmax_maxelem[(((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) / 21)]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ T_reshape) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 2541) {
      int v_ = ((int)(floorf(((max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
    T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_reshape, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 2541) {
    T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[(((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 231) * 924) + (((((((int)blockIdx.x) * 50) + (((int)threadIdx.x) >> 1)) % 462) / 21) * 42)) + (((((((int)blockIdx.x) * 16) + ((int)threadIdx.x)) % 42) / 7) * 7)) + (((((int)blockIdx.x) * 2) + ((int)threadIdx.x)) % 7))];
  }
}

extern "C" __global__ void __launch_bounds__(242) default_function_kernel_1(float* __restrict__ T_reshape, float* __restrict__ T_softmax_maxelem) {
  T_softmax_maxelem[((int)threadIdx.x)] = -3.402823e+38f;
  for (int k = 0; k < 42; ++k) {
    T_softmax_maxelem[((int)threadIdx.x)] = max(T_softmax_maxelem[((int)threadIdx.x)], T_reshape[((((int)threadIdx.x) * 42) + k)]);
  }
}

