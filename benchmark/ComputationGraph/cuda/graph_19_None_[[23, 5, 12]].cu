
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(765) default_function_kernel(float* __restrict__ T_strided_slice, float* __restrict__ ph) {
  if (((int)threadIdx.x) < 168) {
    T_strided_slice[((int)threadIdx.x)] = ph[(((((((int)threadIdx.x) / 21) * 60) + (((((int)threadIdx.x) % 21) / 7) * 12)) + (((int)threadIdx.x) % 7)) + 61)];
  }
}

extern "C" __global__ void __launch_bounds__(168) default_function_kernel_2(float* __restrict__ T_strided_slice, float* __restrict__ compute) {
  compute[((int)threadIdx.x)] = __expf(T_strided_slice[((int)threadIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(765) default_function_kernel_1(float* __restrict__ T_strided_slice) {
  if (((int)threadIdx.x) < 168) {
      int v_ = ((int)(floorf(((max(min(T_strided_slice[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
    T_strided_slice[((int)threadIdx.x)] = max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min(T_strided_slice[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min(T_strided_slice[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min(T_strided_slice[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min(T_strided_slice[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min(T_strided_slice[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min(T_strided_slice[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min(T_strided_slice[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min(T_strided_slice[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_strided_slice[((int)threadIdx.x)], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), T_strided_slice[((int)threadIdx.x)]);
  }
}

