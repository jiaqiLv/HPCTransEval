
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_divide[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(18) default_function_kernel_2(float* __restrict__ T_subtract, float* __restrict__ ph_0) {
  T_subtract[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))] = (__cosf(ph_0[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))]) - ph_0[((((int)blockIdx.x) * 18) + ((int)threadIdx.x))]);
}

