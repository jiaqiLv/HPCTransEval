
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(384) default_function_kernel_3(float* __restrict__ T_strided_slice, float* __restrict__ compute) {
  compute[((int)threadIdx.x)] = __expf(T_strided_slice[((int)threadIdx.x)]);
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_strided_slice, float* __restrict__ ph) {
  if (((int)threadIdx.x) < 384) {
    T_strided_slice[((int)threadIdx.x)] = ph[((((((((int)threadIdx.x) / 192) * 31944) + (((((int)threadIdx.x) % 192) / 48) * 1452)) + (((((int)threadIdx.x) % 48) >> 3) * 44)) + (((int)threadIdx.x) & 7)) + 149194)];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_strided_slice) {
  if (((int)threadIdx.x) < 384) {
    T_strided_slice[((int)threadIdx.x)] = __log10f(T_strided_slice[((int)threadIdx.x)]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_strided_slice) {
  if (((int)threadIdx.x) < 384) {
    T_strided_slice[((int)threadIdx.x)] = roundf(T_strided_slice[((int)threadIdx.x)]);
  }
}

