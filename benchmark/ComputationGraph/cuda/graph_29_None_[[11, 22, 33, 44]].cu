
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(min(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 1.000000e+01f), 0.000000e+00f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = tanhf(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_4(float* __restrict__ T_strided_slice, float* __restrict__ T_transpose) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_strided_slice[(((((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 10) * 1000) + ((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 125) * 100)) + (((((((int)blockIdx.x) * 12) + (((int)threadIdx.x) >> 1)) % 50) / 5) * 10)) + ((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 2)) % 250) / 25))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_strided_slice, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 125) * 31944) + (((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 2)) % 250) / 25) * 1452)) + (((((((int)blockIdx.x) * 12) + (((int)threadIdx.x) >> 1)) % 50) / 5) * 44)) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 10)) + 16973)];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = __log2f(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

