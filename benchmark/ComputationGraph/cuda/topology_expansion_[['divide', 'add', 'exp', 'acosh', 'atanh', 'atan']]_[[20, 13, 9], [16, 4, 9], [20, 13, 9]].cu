
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel_2(float* __restrict__ T_add, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  T_add[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] + ph_3[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = atanf((ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = acoshf(__expf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanhf(__expf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

