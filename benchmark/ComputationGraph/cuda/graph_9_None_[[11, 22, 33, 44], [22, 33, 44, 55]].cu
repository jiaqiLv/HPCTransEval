
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = min(9.000000e+00f, T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_strided_slice, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 125) * 31944) + (((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 2)) % 250) / 25) * 1452)) + (((((((int)blockIdx.x) * 12) + (((int)threadIdx.x) >> 1)) % 50) / 5) * 44)) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 10)) + 16973)];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_5(float* __restrict__ T_fast_tanh, float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_fast_tanh[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * (((T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * -2.760768e-16f) + 2.000188e-13f)) + -8.604672e-11f)) + 5.122297e-08f)) + 1.485722e-05f)) + 6.372619e-04f)) + 4.893525e-03f)) / (((T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * 1.198258e-06f) + 1.185347e-04f)) + 2.268435e-03f)) + 4.893525e-03f));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_4(float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(-9.000000e+00f, T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_strided_slice, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 125) * 79860) + (((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 2)) % 250) / 25) * 2420)) + (((((((int)blockIdx.x) * 12) + (((int)threadIdx.x) >> 1)) % 50) / 5) * 55)) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 10)) + 933559)];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_strided_slice, float* __restrict__ T_strided_slice_1) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] - T_strided_slice_1[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

