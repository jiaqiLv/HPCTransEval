
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(16) default_function_kernel(float* __restrict__ T_divide, float* __restrict__ ph_0, float* __restrict__ ph_3) {
  if (((((int)blockIdx.x) * 2) + (((int)threadIdx.x) >> 3)) < 9) {
    T_divide[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] = (ph_0[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))] / ph_3[((((int)blockIdx.x) * 16) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((int)blockIdx.x)] = __expf((ph_0[((int)blockIdx.x)] - asinhf(ph_0[((int)blockIdx.x)])));
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 3)) < 9) {
    compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = fabsf((ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] - asinhf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))])));
  }
}

