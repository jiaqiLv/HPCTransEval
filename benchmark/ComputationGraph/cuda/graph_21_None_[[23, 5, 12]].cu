
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_5(float* __restrict__ T_expand_dims, float* __restrict__ T_reverse_sequence, float* __restrict__ compute) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3795) {
    T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] - T_expand_dims[(((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3)]) - __logf(compute[(((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3)]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_reverse_sequence, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 345) {
    T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[(((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 15) * 60) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 12)) + 48) - (((((((int)threadIdx.x) >> 2) + ((int)blockIdx.x)) % 15) / 3) * 12))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_expand_dims, float* __restrict__ T_reverse_sequence) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3795) {
    T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_expand_dims[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) % 1380)];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_expand_dims, float* __restrict__ T_reverse_sequence) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 345) {
    T_expand_dims[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_reverse_sequence[(((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 15) * 60) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 60))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_6(float* __restrict__ T_fast_exp, float* __restrict__ T_reverse_sequence) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 3795) {
      int v_ = ((int)(floorf(((max(min(T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
    T_fast_exp[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min(T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min(T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min(T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min(T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min(T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min(T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min(T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min(T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), T_reverse_sequence[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_4(float* __restrict__ T_expand_dims, float* __restrict__ T_reverse_sequence, float* __restrict__ compute) {
  if (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) < 1265) {
    compute[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k = 0; k < 12; ++k) {
    if (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) < 1265) {
      compute[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (compute[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + __expf((T_reverse_sequence[(((((int)blockIdx.x) * 12288) + (((int)threadIdx.x) * 12)) + k)] - T_expand_dims[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))])));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ T_expand_dims, float* __restrict__ T_reverse_sequence) {
  if (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) < 1265) {
    T_expand_dims[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int k = 0; k < 12; ++k) {
    if (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) < 1265) {
      T_expand_dims[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(T_expand_dims[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], T_reverse_sequence[(((((int)blockIdx.x) * 12288) + (((int)threadIdx.x) * 12)) + k)]);
    }
  }
}

