
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_5(float* __restrict__ T_softmax_maxelem, float* __restrict__ resize) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 8)) < 625) {
    resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] - T_softmax_maxelem[(((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) / 625) * 2500) + (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) % 2500))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ ph, float* __restrict__ resize) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 8)) < 625) {
    resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((((ph[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 625) * 1024) + (max(min(((int)floorf((((((float)((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 1250) / 25)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f))), 31), 0) * 32)) + max(min(((int)floorf((((((float)(((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 50)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f))), 31), 0))] * (1.000000e+00f - ((((((float)(((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 50)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f) - ((float)((int)floorf((((((float)(((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 50)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f))))))) + (ph[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 625) * 1024) + (max(min(((int)floorf((((((float)((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 1250) / 25)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f))), 31), 0) * 32)) + max(min((((int)floorf((((((float)(((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 50)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f))) + 1), 31), 0))] * ((((((float)(((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 50)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f) - ((float)((int)floorf((((((float)(((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 50)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f))))))) * (1.000000e+00f - ((((((float)((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 1250) / 25)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f) - ((float)((int)floorf((((((float)((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 1250) / 25)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f))))))) + (((ph[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 625) * 1024) + (max(min((((int)floorf((((((float)((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 1250) / 25)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f))) + 1), 31), 0) * 32)) + max(min(((int)floorf((((((float)(((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 50)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f))), 31), 0))] * (1.000000e+00f - ((((((float)(((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 50)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f) - ((float)((int)floorf((((((float)(((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 50)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f))))))) + (ph[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 625) * 1024) + (max(min((((int)floorf((((((float)((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 1250) / 25)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f))) + 1), 31), 0) * 32)) + max(min((((int)floorf((((((float)(((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 50)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f))) + 1), 31), 0))] * ((((((float)(((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 50)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f) - ((float)((int)floorf((((((float)(((((int)blockIdx.x) * 24) + ((int)threadIdx.x)) % 50)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f))))))) * ((((((float)((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 1250) / 25)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f) - ((float)((int)floorf((((((float)((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) % 1250) / 25)) + 5.000000e-01f) * 6.400000e-01f) - 5.000000e-01f)))))));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_6(float* __restrict__ resize) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 8)) < 625) {
      int v_ = ((int)(floorf(((max(min(resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) + 1.270000e+02f)) << 23;
    resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(((*(float *)(&(v_))) * ((((((((((((((1.987569e-04f * (max(min(resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.398200e-03f) * (max(min(resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 8.333452e-03f) * (max(min(resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 4.166580e-02f) * (max(min(resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.666667e-01f) * (max(min(resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 5.000000e-01f) * (max(min(resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) * (max(min(resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + (max(min(resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) - (floorf(((max(min(resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 8.837627e+01f), -8.837626e+01f) * 1.442695e+00f) + 5.000000e-01f)) * 6.931472e-01f))) + 1.000000e+00f)), resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_8(float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_norm, float* __restrict__ resize) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 8)) < 625) {
    T_softmax_norm[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] / T_softmax_maxelem[(((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) / 625) * 2500) + (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) % 2500))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ resize) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 8)) < 625) {
    resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((0.000000e+00f < resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) ? resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] : (resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * 5.000000e-01f));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_4(float* __restrict__ T_softmax_maxelem, float* __restrict__ resize) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_softmax_maxelem[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int k = 0; k < 16; ++k) {
    if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
      T_softmax_maxelem[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(T_softmax_maxelem[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], resize[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 625) * 40000) + (k * 2500)) + (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) % 2500))]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_7(float* __restrict__ T_softmax_maxelem, float* __restrict__ resize) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_softmax_maxelem[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k = 0; k < 16; ++k) {
    if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
      T_softmax_maxelem[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_softmax_maxelem[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + resize[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 625) * 40000) + (k * 2500)) + (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) % 2500))]);
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ resize) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 8)) < 625) {
    resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * 7.692308e-02f);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ resize) {
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 8)) < 625) {
    resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (resize[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + 3.000000e+00f);
  }
}

