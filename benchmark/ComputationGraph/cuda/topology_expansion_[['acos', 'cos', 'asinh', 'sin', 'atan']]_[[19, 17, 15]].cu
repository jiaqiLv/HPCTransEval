
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(4) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))] = acosf(ph_0[((((int)blockIdx.x) * 4) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_1(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = asinhf(__cosf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]));
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_2(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))] = __sinf(ph_0[((((int)blockIdx.x) * 8) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(32) default_function_kernel_3(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))] = atanf(ph_0[((((int)blockIdx.x) * 32) + ((int)threadIdx.x))]);
}

