
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_concat, float* __restrict__ T_strided_slice, float* __restrict__ T_strided_slice_1) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 1125) {
    T_concat[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((1125 <= ((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3))) ? T_strided_slice[(((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) - 9000)] : T_strided_slice_1[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_strided_slice, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 1125) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 225) * 31944) + (((((((int)blockIdx.x) * 62) + (((int)threadIdx.x) >> 1)) % 450) / 45) * 1452)) + (((((((int)blockIdx.x) * 34) + ((int)threadIdx.x)) % 90) / 9) * 44)) + (((((int)blockIdx.x) * 7) + ((int)threadIdx.x)) % 9)) + 16973)];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ T_concat, float* __restrict__ T_strided_slice) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 1125) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_concat[(((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 225) * 900) + (((((((int)blockIdx.x) * 62) + (((int)threadIdx.x) >> 1)) % 450) / 45) * 90)) + (((((((int)blockIdx.x) * 34) + ((int)threadIdx.x)) % 90) / 9) * 9)) + (((((int)blockIdx.x) * 7) + ((int)threadIdx.x)) % 9))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_strided_slice, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 1125) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 225) * 31944) + (((((((int)blockIdx.x) * 62) + (((int)threadIdx.x) >> 1)) % 450) / 45) * 1452)) + (((((((int)blockIdx.x) * 34) + ((int)threadIdx.x)) % 90) / 9) * 44)) + (((((int)blockIdx.x) * 7) + ((int)threadIdx.x)) % 9)) + 50414)];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_7(float* __restrict__ T_strided_slice, float* __restrict__ T_transpose) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 1125) {
    T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_strided_slice[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 225) * 900) + ((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 10) * 90)) + ((((((int)blockIdx.x) * 62) + (((int)threadIdx.x) >> 1)) % 450) / 5))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_6(float* __restrict__ T_concat, float* __restrict__ T_strided_slice, float* __restrict__ T_strided_slice_1) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 1125) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] - T_strided_slice_1[(((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 225) * 90) + (((((int)blockIdx.x) * 34) + ((int)threadIdx.x)) % 90))]) - __logf(T_concat[(((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 225) * 90) + (((((int)blockIdx.x) * 34) + ((int)threadIdx.x)) % 90))]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_5(float* __restrict__ T_concat, float* __restrict__ T_strided_slice, float* __restrict__ T_strided_slice_1) {
  if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 225) {
    T_concat[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = 0.000000e+00f;
  }
  for (int k = 0; k < 10; ++k) {
    if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 225) {
      T_concat[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_concat[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + __expf((T_strided_slice[((((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) / 45) * 900) + (k * 90)) + (((((int)blockIdx.x) * 34) + ((int)threadIdx.x)) % 90))] - T_strided_slice_1[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))])));
    }
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_4(float* __restrict__ T_strided_slice, float* __restrict__ T_strided_slice_1) {
  if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 225) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = -3.402823e+38f;
  }
  for (int k = 0; k < 10; ++k) {
    if (((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) < 225) {
      T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], T_strided_slice_1[((((((((int)blockIdx.x) * 512) + (((int)threadIdx.x) >> 1)) / 45) * 900) + (k * 90)) + (((((int)blockIdx.x) * 34) + ((int)threadIdx.x)) % 90))]);
    }
  }
}

