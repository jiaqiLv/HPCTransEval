
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1000) default_function_kernel_3(float* __restrict__ T_strided_slice, float* __restrict__ T_strided_slice_1) {
  T_strided_slice[((int)threadIdx.x)] = -3.402823e+38f;
  for (int k = 0; k < 10; ++k) {
    T_strided_slice[((int)threadIdx.x)] = max(T_strided_slice[((int)threadIdx.x)], T_strided_slice_1[((((((int)threadIdx.x) / 10) * 100) + (k * 10)) + (((int)threadIdx.x) % 10))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_strided_slice, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 125) * 79860) + (((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 2)) % 250) / 25) * 2420)) + (((((((int)blockIdx.x) * 12) + (((int)threadIdx.x) >> 1)) % 50) / 5) * 55)) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 10)) + 933559)];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_strided_slice, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((((((((int)blockIdx.x) * 128) + (((int)threadIdx.x) >> 3)) / 125) * 31944) + (((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 2)) % 250) / 25) * 1452)) + (((((((int)blockIdx.x) * 12) + (((int)threadIdx.x) >> 1)) % 50) / 5) * 44)) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 10)) + 16973)];
  }
}

extern "C" __global__ void __launch_bounds__(1000) default_function_kernel_4(float* __restrict__ T_strided_slice, float* __restrict__ T_strided_slice_1, float* __restrict__ compute) {
  compute[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k = 0; k < 10; ++k) {
    compute[((int)threadIdx.x)] = (compute[((int)threadIdx.x)] + __expf((T_strided_slice[((((((int)threadIdx.x) / 10) * 100) + (k * 10)) + (((int)threadIdx.x) % 10))] - T_strided_slice_1[((int)threadIdx.x)])));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_5(float* __restrict__ T_strided_slice, float* __restrict__ T_strided_slice_1, float* __restrict__ compute, float* __restrict__ compute_1) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    compute[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] - T_strided_slice_1[(((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 10) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 10))]) - __logf(compute_1[(((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 25) * 10) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 10))]));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_strided_slice, float* __restrict__ T_strided_slice_1) {
  if (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) >> 4)) < 625) {
    T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (T_strided_slice[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + T_strided_slice_1[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

