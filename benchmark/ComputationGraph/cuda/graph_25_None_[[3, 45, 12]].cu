
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_transpose) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 405) {
    T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = floorf(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_transpose, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 405) {
    T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[((((((int)blockIdx.x) + ((int)threadIdx.x)) % 3) * 540) + (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) / 3))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_transpose, float* __restrict__ compute) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 405) {
    compute[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(min(T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))], 1.111000e+01f), 1.100000e+00f);
  }
}

