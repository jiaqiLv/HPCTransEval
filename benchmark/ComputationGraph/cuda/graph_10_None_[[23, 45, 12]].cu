
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_5(float* __restrict__ T_reshape, float* __restrict__ T_transpose) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 1035) {
    T_transpose[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_reshape[(((((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) % 1380) * 3) + (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 345))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_reshape, float* __restrict__ ph) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 1035) {
    T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ph[(((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 3) * 12) + (((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 12))];
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_3(float* __restrict__ T_reshape) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 1035) {
    T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * (((T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * -2.760768e-16f) + 2.000188e-13f)) + -8.604672e-11f)) + 5.122297e-08f)) + 1.485722e-05f)) + 6.372619e-04f)) + 4.893525e-03f)) / (((T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * (((T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] * T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) * 1.198258e-06f) + 1.185347e-04f)) + 2.268435e-03f)) + 4.893525e-03f));
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_1(float* __restrict__ T_reshape) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 1035) {
    T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = min(9.000000e+00f, T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_2(float* __restrict__ T_reshape) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 1035) {
    T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = max(-9.000000e+00f, T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel_4(float* __restrict__ T_reshape, float* __restrict__ T_reshape_1) {
  if (((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) < 1035) {
    T_reshape[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = T_reshape_1[((((((((int)blockIdx.x) * 256) + (((int)threadIdx.x) >> 2)) / 15) * 60) + (((((((int)blockIdx.x) * 4) + ((int)threadIdx.x)) % 60) / 3) * 3)) + ((((int)blockIdx.x) + ((int)threadIdx.x)) % 3))];
  }
}

