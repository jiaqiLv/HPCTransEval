
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(8) default_function_kernel_2(float* __restrict__ T_reverse_sequence, float* __restrict__ T_softmax_maxelem) {
  T_softmax_maxelem[((int)threadIdx.x)] = -3.402823e+38f;
  for (int k = 0; k < 21; ++k) {
    T_softmax_maxelem[((int)threadIdx.x)] = max(T_softmax_maxelem[((int)threadIdx.x)], T_reverse_sequence[((((int)threadIdx.x) * 21) + k)]);
  }
}

extern "C" __global__ void __launch_bounds__(8) default_function_kernel_4(float* __restrict__ T_reverse_sequence, float* __restrict__ T_softmax_maxelem) {
  T_softmax_maxelem[((int)threadIdx.x)] = 0.000000e+00f;
  for (int k = 0; k < 21; ++k) {
    T_softmax_maxelem[((int)threadIdx.x)] = (T_softmax_maxelem[((int)threadIdx.x)] + T_reverse_sequence[((((int)threadIdx.x) * 21) + k)]);
  }
}

extern "C" __global__ void __launch_bounds__(765) default_function_kernel_1(float* __restrict__ T_reverse_sequence, float* __restrict__ T_strided_slice) {
  if (((int)threadIdx.x) < 168) {
    T_reverse_sequence[((int)threadIdx.x)] = T_strided_slice[(((((((int)threadIdx.x) / 21) * 21) + (((int)threadIdx.x) % 7)) + 14) - (((((int)threadIdx.x) % 21) / 7) * 7))];
  }
}

extern "C" __global__ void __launch_bounds__(168) default_function_kernel_3(float* __restrict__ T_reverse_sequence, float* __restrict__ T_softmax_maxelem) {
  T_reverse_sequence[((int)threadIdx.x)] = __expf((T_reverse_sequence[((int)threadIdx.x)] - T_softmax_maxelem[(((int)threadIdx.x) / 21)]));
}

extern "C" __global__ void __launch_bounds__(765) default_function_kernel(float* __restrict__ T_strided_slice, float* __restrict__ ph) {
  if (((int)threadIdx.x) < 168) {
    T_strided_slice[((int)threadIdx.x)] = ph[(((((((int)threadIdx.x) / 21) * 60) + (((((int)threadIdx.x) % 21) / 7) * 12)) + (((int)threadIdx.x) % 7)) + 85)];
  }
}

extern "C" __global__ void __launch_bounds__(168) default_function_kernel_5(float* __restrict__ T_reverse_sequence, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_norm) {
  T_softmax_norm[((int)threadIdx.x)] = (T_reverse_sequence[((int)threadIdx.x)] / T_softmax_maxelem[(((int)threadIdx.x) / 21)]);
}

