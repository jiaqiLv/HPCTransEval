
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(640) default_function_kernel_1(float* __restrict__ T_broadcast_to, float* __restrict__ ph) {
  T_broadcast_to[((int)threadIdx.x)] = ((0.000000e+00f < T_broadcast_to[((int)threadIdx.x)]) ? T_broadcast_to[((int)threadIdx.x)] : (T_broadcast_to[((int)threadIdx.x)] * ph[((((int)threadIdx.x) % 160) / 5)]));
}

extern "C" __global__ void __launch_bounds__(640) default_function_kernel_2(float* __restrict__ T_broadcast_to, float* __restrict__ T_fast_erf) {
  T_fast_erf[((int)threadIdx.x)] = ((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * -2.726142e-10f) + 2.770681e-08f)) + -2.101024e-06f)) + -5.692506e-05f)) + -7.349906e-04f)) + -2.954600e-03f)) + -1.609603e-02f)) / (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * (((max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f) * max(min(T_broadcast_to[((int)threadIdx.x)], 4.000000e+00f), -4.000000e+00f)) * -1.456607e-05f) + -2.133740e-04f)) + -1.682827e-03f)) + -7.373329e-03f)) + -1.426474e-02f));
}

extern "C" __global__ void __launch_bounds__(1024) default_function_kernel(float* __restrict__ T_broadcast_to, float* __restrict__ ph) {
  if (((int)threadIdx.x) < 640) {
    T_broadcast_to[((int)threadIdx.x)] = ph[(((((((int)threadIdx.x) % 160) / 5) * 12) + (((int)threadIdx.x) % 5)) + 14)];
  }
}

