
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(64) default_function_kernel_1(float* __restrict__ T_mod, float* __restrict__ ph_0) {
  T_mod[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))] = fmodf(__sinf(ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]), ph_0[((((int)blockIdx.x) * 64) + ((int)threadIdx.x))]);
}

extern "C" __global__ void __launch_bounds__(12) default_function_kernel(float* __restrict__ compute, float* __restrict__ ph_0) {
  compute[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))] = asinhf(ph_0[((((int)blockIdx.x) * 12) + ((int)threadIdx.x))]);
}

